#include "hip/hip_runtime.h"
/*
   Copyright 2022 Cesar Miranda Meza (alias: Mortrack)

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/
#include "CenyMLdeepLearning_SG.h"
#include "../../../../CenyML_library_skeleton/otherLibraries/cuda/CUDA_check.h"
// IMPORTANT NOTE: This library uses the math.h library and therefore, remember
// 		to use the "-lm" flag when compiling it.


/**
* The "getSingleNeuronDNN_singleGPU()" function is used to apply the machine
* learning algorithm called single neuron in Deep Neural Network as formulated
* in the master thesis of Cesar Miranda Meza called "Machine learning to support
* applications with embedded systems and parallel computing", but in its single
* GPU parallel version. Within this process, the best fitting equation with the
* form of "y_hat = b_0 + w_1*x_1 + w_2*x_2 + ... + w_m*x_m" will be identified
* with respect to the sampled data given through the argument pointer variables
* "neuron->X" and "neuron->Y". As a result, the identified coefficient values
* will be stored in the argument pointer variable "neuron->w_new". With respect
* to the struct pointer variable "neuron", it should contain all the information
* required in order to be able to create and make an artificial neuron. Its
* accessible inner elements will be described in the following list:
*
*
* @param int gpuDevice - This argument will represent the desired GPU (device)
* 			with which the implementer wants this algorithm to be
* 			parallelized.
*
* @param double *X - This argument will contain the pointer to a memory
* 		allocated input matrix, from which the desired machine learning
* 		algorithm will be calculated. THIS VARIABLE SHOULD BE ALLOCATED
*		AND INITIALIZED BEFORE CALLING THIS FUNCTION WITH A SIZE OF "n"
* 		TIMES "m" 'DOUBLE' MEMORY SPACES.
*
* @param double *w_first - This argument will contain the pointer to a memory
* 			allocated coefficient matrix. The use of this variable
* 			will difer depending on the value assigned in the
*			argument variable "isInitial_w", whose possible outcomes
* 			are listed below:
*			1) "isInitial_w"=(int)1 --> "w_first"
*			HAS TO BE INITIALIZED BEFORE CALLING THIS FUNCTION
* 			because its defined coefficient values will be assigned
*			to the neuron as its initial weight values before
* 			starting its training process.
*			2) "isInitial_w"=(int)0 --> "w_first"
*			does not require to be initialized but has to be
* 			allocated in memory. After this function concludes its
* 			processes, the implementer will be able to know what were
* 			the initial weight values that the neuron had when it was
*			created. Regardless of the value of "isInitial_w",
* 			"w_first" SHOULD BE ALLOCATED BEFORE CALLING THIS
* 			FUNCTION WITH A SIZE OF "1" TIMES "m+1" 'DOUBLE' MEMORY
* 			SPACES.
*
* @param double *Y - This argument will contain the pointer to a memory
* 		allocated output matrix, representing the real data of the
* 		system under study. This variable will be used as a reference to
* 		apply the desired machine learning algorithm. THIS VARIABLE
* 		SHOULD BE ALLOCATED AND INITIALIZED BEFORE CALLING THIS FUNCTION
* 		WITH A SIZE OF "n" TIMES "p=1" 'DOUBLE' MEMORY SPACES.
*
* @param int n - This argument will represent the total number of samples (rows)
* 		that the input matrix has, with which the output data was
* 		obtained.
*
* @param int m - This argument will represent the total number of features
* 		(independent variables) that the input matrix has, with which
* 		the output data was obtained.
*
* @param int p - This argument will represent the total number of outputs that
* 		exist in the the output matrix, containing the real results of
* 		the system under study.
*
* @param char isInitial_w = This argument variable will work as a flag to
* 			indicate whether the coefficients contained in the
* 			argument variable "w_first" will be used as the initial
* 			weight values for the neuron to be created or not. The
* 			possible values for "isInitial_w" are the following:
*			1) "isInitial_w"=(int)1 --> the coefficient values of
* 			"w_first" will be assigned to the neuron as its initial
* 			weight values before starting its training process.
*			2) "isInitial_w"=(int)0 --> the coefficient values of
* 			"w_first" will not be assigned to the neuron as its
* 			initial weight values and after having called this
* 			function, the implementer will be able to retrieve from
* 			"w_first" the coefficient values with which the neuron
*			had been created before starting its learning process.
*
* @param char isClassification = This argument variable will work as a flag to
* 				indicate to the neron if it is expected from it
* 				to interpret the given data of "X" and "Y" as if
* 				their were meant for a classification problem or
* 				not. The possible valid values for this flag are
* 				the following:
*				1) "isClassification" = (int) 1 --> The neuron
* 				will interpret the data of "X" and "Y" as if they
* 				were meant for a classification problem.
*				2) "isClassification" = (int) 0 --> The neuron
* 				will interpret the data of "X" and "Y" as if they
* 				were meant for a regression problem.
*
* @param double threshold - This argument will represent desired threshold that
* 			the implementer desired the neuron to consider in
* 			classification problems. In this regard, whenever the
* 			predicted output of the neuron is higher than the
* 			defined threshold value, then that prediction should be
*			interpreted as group 1 (ussually refered to as the binary
* 			output 1). Conversely, if the predicted value is lower
* 			than the defined threshold value, then that prediction
* 			should be interpreted as group 2 (ussually refered to as
*			the binary output 0). However, have in mind that
* 			"threshold" will only be used by the neuron if the
* 			argument variable "isClassification" = 1.
*
* @param int desiredValueForGroup1 - This argument will represent the desired
*				label value to whenever an output of the neuron
* 				predicts the classification group 1. Ussually,
* 				this is label with the value of "(int) 1" but any
* 				other customized value can be assigned by the
* 				implementer. However, have in mind that this
* 				argument variable will be considered by the
* 				neuron as long as the argument variable
*				"isClassification" = 1 and only when the
*				implementer requests to the neuron a prediction
* 				through the function
* 				"predictSingleNeuronDNN_singleGPU()".
*
* @param int desiredValueForGroup2 - This argument will represent the desired
*				label value to whenever an output of the neuron
* 				predicts the classification group 2. Ussually,
* 				this is label with the value of "(int) -1" but
* 				any other customized value can be assigned by the
* 				implementer. However, have in mind that this
* 				argument variable will be considered by the
* 				neuron as long as the argument variable
*				"isClassification" = 1 and only when the
*				implementer requests to the neuron a prediction
* 				through the function
* 				"predictSingleNeuronDNN_singleGPU()".
*
* @param int activationFunctionToBeUsed - This argument will represent the
* 					identifier of the desired activation
* 					function to be used by the neuron during
* 					its training process. Its possible valid
* 					values are the following:
*					0 = Rectified Linear Units (ReLU).
*					1 = Hyperbolic tangent (tanh).
*					2 = Logistic function.
*					3 = Raise to the 1st power.
*					4 = Raise to the 2nd power.
*					5 = Raise to the 3rd power.
*					6 = Raise to the 4th power.
*					7 = Raise to the 5th power.
*					8 = Raise to the 6th power.
*					9 = 1st order degree exponential.
*					10 = 2nd order degree exponential.
*
* @param double learningRate - This argument will represent the hyperparameter
* 			value known as the learning rate of the artificial
* 			neuron. Note that there is no way to know what is going
* 			to be the best learning rate value for your particular
* 			problem to be solved by the neuron because the best one
* 			differs from one problem to another. Therefore, you will
* 			most likely have to experiment with several values until
* 			you find the model solution that satisfies you the most.
*
* @param double stopAboveThisAccuracy - This argument will represent a a stop
* 				value for the training process. The way this
* 				value will work is that if the neuron gets an
* 				evaluation metric result that is strictly higher
* 				than the one defined in "stopAboveThisAccuracy",
* 				then the neuron will stop its training process
*				and this function will end. Note that the
* 				evaluation metric to be used will be the adjusted
* 				R squared regardless if the data is for
* 				classification or not.
*
* @param int maxEpochs - This argument will represent the maximum number of
* 			epochs that are desired for the training process of the
*			artificial neuron. Note that for each epoch that occurs,
*			that should be interpreted as the neuron having updated
*			its weight values one time.
*
* @param char isReportLearningProgress = This argument variable will work as a
*				flag to indicate to the neuron if it is desired
* 				that it reports its learning progress to the
* 				user. The following will list the possible valid
* 				outcomes for this variable:
*				1) "isReportLearningProgress" = (int) 1:
*				The neuron will interpret this as being
* 				instructed to report its learning progress to the
* 				user through the window terminal by displaying
*				messages over time.
*				2) "isReportLearningProgress" = (int) 0:
*				The neuron will interpret this as being
* 				instructed not to report its learning progress.
*
* @param int reportEachSpecifiedEpochs - This argument variable will indicate
*				how many each amount of epochs it is desired by
* 				the implementer that the artificial neuron
* 				reports its learning progress to the user.
* 				However, in order for the neuron to consider this
* 				variable, it will be strictly needed to set the
*				argument variable "isReportLearningProgress" =
* 				(int) 1.
*
* @param double *w_best - This argument will contain the pointer to a memory
*			allocated variable in which we will store the identified
* 			best fitting coefficient values for the model of a single
* 			neuron in Deep Neural Network. These coefficients will
* 			each be stored in the same row but under different
* 			columns where the first coefficient (b_0) will be stored
* 			in the column with index 0; the second coefficient (w_1)
* 			will be stored in the column index 1 and; the last
* 			coefficient (w_m) will be stored in the column index m.
* 			IT IS INDISPENSABLE THAT THIS VARIABLE IS ALLOCATED
* 			BEFORE CALLING THIS FUNCTION WITH A VARIABLE SIZE OF "1"
* 			TIMES "m+1" 'DOUBLE' MEMORY SPACES.
*
* @param double bestAccuracy - This argument will contain the value of the best
*			accuracy that the neuron was able to achieve during its
* 			training process.
*
* @param double *w_new - This argument will contain the pointer to a memory
*			allocated variable in which we will store the last
*			identified coefficient values for the model of a single
* 			neuron in Deep Neural Network. These coefficients will
* 			each be stored in the same row but under different
* 			columns where the first coefficient (b_0) will be stored
* 			in the column with index 0; the second coefficient (w_1)
* 			will be stored in the column index 1 and; the last
* 			coefficient (w_m) will be stored in the column index m.
* 			IT IS INDISPENSABLE THAT THIS VARIABLE IS ALLOCATED
* 			BEFORE CALLING THIS FUNCTION WITH A VARIABLE SIZE OF "1"
* 			TIMES "m+1" 'DOUBLE' MEMORY SPACES.
*
* NOTE: RESULT IS STORED IN THE MEMORY ALLOCATED POINTER VARIABLE "w_best" that
* 	is contained in the struct pointer variable "neuron".
*
* @return void
*
* @author Miranda Meza Cesar
* CREATION DATE: JANUARY 24, 2022
* LAST UPDATE: JANUARY 25, 2022
*/
void getSingleNeuronDNN_singleGPU(struct singleNeuronDnnStruct_singleGPU *neuron) {
	// If the requested GPU (device) is less than zero, then emit an error message and terminate the program. Otherwise, continue with the program.
	if (neuron->gpuDevice < 0) {
		printf("\nERROR: The identifier of the requested GPU (device) must be equal or greater than 0.\n");
		exit(1);
	}
	// If the value of "neuron->maxUnrollingLoop" is not in the range of 1 and 10, then emit an error message and terminate the program. Otherwise, continue with the program.
	if ((neuron->maxUnrollingLoop<1) && (neuron->maxUnrollingLoop>10)) {
		printf("\nERROR: The defined value for \"maxUnrollingLoop\" in the struct of \"singleNeuronDnnStruct\" that you created can only have a whole value in the range of 1 and 10. Please add a valid value to it.\n");
		exit(1);
	}
	// If the machine learning samples are less than value of two, then emit an error message and terminate the program. Otherwise, continue with the program.
	if (neuron->n < 2) {
		printf("\nERROR: The machine learning samples must be equal or greater than 2 for this particular algorithm.\n");
		exit(1);
	}
	// If the machine learning features are less than the value of one, then emit an error message and terminate the program. Otherwise, continue with the program.
	if (neuron->m < 1) {
		printf("\nERROR: The machine learning features (independent variables) must be equal or greater than 1 for this particular algorithm.\n");
		exit(1);
	}
	// If the output of the system under study is different than the value of one, then emit an error message and terminate the program. Otherwise, continue with the program.
	if (neuron->p != 1) {
		printf("\nERROR: The outputs of the system under study must be equal to 1 for this particular algorithm.\n");
		exit(1);
	}
	// If the identifier assigned to "neuron->activationFunctionToBeUsed" is not in the range of 0 and 11, then emit an error message and terminate the program. Otherwise, continue with the program.
	if ((neuron->activationFunctionToBeUsed>11) && (neuron->activationFunctionToBeUsed<0)) {
		printf("\nERROR: The defined activation function identifier assigned to \"activationFunctionToBeUsed\" in the struct of \"singleNeuronDnnStruct\" that you created must be a whole value in the range of 0 to 11. Please add a valid identifier number to it.\n");
		exit(1);
	}
	// If the flag "neuron->isClassification" has a value different of 0 and 1, then emit an error message and terminate the program. Otherwise, continue with the program.
	if ((neuron->isClassification!=0) && (neuron->isClassification!=1)) {
		printf("\nERROR: The defined value for the flag \"isClassification\" in the struct of \"singleNeuronDnnStruct\" that you created can only have a value of either 0 or 1. Please add a valid value to it.\n");
		exit(1);
	}
	// If the flag "neuron->isInitial_w" has a value different of 0 and 1, then emit an error message and terminate the program. Otherwise, continue with the program.
	if ((neuron->isInitial_w!=1) && (neuron->isInitial_w!=0)) {
		printf("\nERROR: The defined value for the flag \"isInitial_w\" in the struct of \"singleNeuronDnnStruct\" that you created can only have a value of either 0 or 1. Please add a valid value to it.\n");
		exit(1);
	}
	// If the flag "neuron->isReportLearningProgress" has a value different of 0 and 1, then emit an error message and terminate the program. Otherwise, continue with the program.
	if ((neuron->isReportLearningProgress!=1) && (neuron->isReportLearningProgress!=0)) {
		printf("\nERROR: The defined value for the flag \"isReportLearningProgress\" in the struct of \"singleNeuronDnnStruct\" that you created can only have a value of either 0 or 1. Please add a valid value to it.\n");
		exit(1);
	}
	// If the requested epochs are less than the value of one, then emit an error message and terminate the program. Otherwise, continue with the program.
	if (neuron->maxEpochs < 1) {
		printf("\nERROR: The defined value for \"maxEpochs\" in the struct of \"singleNeuronDnnStruct\" that you created must be equal or greater than 1 for this particular algorithm. Please add a valid value to it.\n");
		exit(1);
	}
	// If the "neuron->reportEachSpecifiedEpochs" is less than one and greater than "neuron->maxEpochs", then emit an error message and terminate the program. Otherwise, continue with the program.
	if ((neuron->reportEachSpecifiedEpochs<1) && (neuron->maxEpochs<neuron->reportEachSpecifiedEpochs)) {
		printf("\nERROR: The defined value for \"reportEachSpecifiedEpochs\" in the struct of \"singleNeuronDnnStruct\" that you created cannot be less than 1 and cannot be greater than the value of \"maxEpochs\" contained in such struct. Please add a valid value to it.\n");
		exit(1);
	}
	// If the value of "neuron->stopAboveThisAccuracy" is not in the range of 0 and 1, then emit an error message and terminate the program. Otherwise, continue with the program.
	if ((neuron->stopAboveThisAccuracy<0) && (neuron->stopAboveThisAccuracy>1)) {
		printf("\nERROR: The defined value for the flag \"stopAboveThisAccuracy\" in the struct of \"singleNeuronDnnStruct\" that you created can only have a value in the range of 0 and 1. Please add a valid value to it.\n");
		exit(1);
	}
	
	
	// ------- SELECTION AND INITIALIZATION OF THE DESIRED GPU ------- //
	// We select the desired GPU by the implementer and inform in the terminal the name of such GPU.
	hipDeviceProp_t gpuProperties;
	CHECK(hipGetDeviceProperties(&gpuProperties, neuron->gpuDevice)); // We obtain the details of the GPU that was defined by the implementer.
	printf("\nThe GPU (device) %d: %s, has been selected by the CenyML library.\n", neuron->gpuDevice, gpuProperties.name);
	CHECK(hipSetDevice(neuron->gpuDevice)); // We select the GPU that was requested by the implementer.
	
	// Set up the execution configurations that will be assigned to the selected GPU.
	dim3 block_32x_1y(32, 1); // We define the number of GPU threads per block.
	dim3 grid_n((neuron->n + block_32x_1y.x - 1) / block_32x_1y.x, 1); // We define the number of blocks that our GPU will manage when the number of samples is "neuron->n".
	
	// We determine what number of Unrolling Loop strategy will be applied according to the number of samples and the value of "neuron->maxUnrollingLoop" that were given by the implementer/user.
	int numberOfUnrollingLoop1; // Variable used to store the number of unrolling loops that the algorithm will use in the first Unrolling Parallel Reduction strategy that will be applied for each case.
	for (numberOfUnrollingLoop1=neuron->maxUnrollingLoop; numberOfUnrollingLoop1>0; numberOfUnrollingLoop1--) {
		// NOTE: The idea in this for-loop is to find the highest number, up to a maximum of 10, that can completely divide the number
		//	  of blocks defined for the selected GPU. However, because the process of defining the number of blocks is conveniently
		//	  automated for performance purposes, the implementer can attempt to achieve the highest or a higher unrolling loop if
		//	  he changes the number of input samples given to this function.
		if (grid_n.x%numberOfUnrollingLoop1 == 0) {
			if (numberOfUnrollingLoop1 == 1) {
				printf("This algorithm WILL NOT apply the \"Unrolling Loop Strategy\" due to the number of samples given and/or the defined maximum unrolling loop.\n");
			} else {
				printf("This algorithm will apply the \"Unrolling%d Loop Strategy\" for each case applicable (the current maximum limit is %d).\n", numberOfUnrollingLoop1, neuron->maxUnrollingLoop);
			}
			break;
		}
	}
	int trueUnrollingSize1 = grid_n.x/numberOfUnrollingLoop1; // This variable is used to store the grid size that will be considered for all the processes that apply the first Unrolling Parallel Reduction strategy for each case, for performance purposes.
	
	// We configure the shared memory of the current GPU.
	hipSharedMemConfig pConfig = hipSharedMemBankSizeEightByte; // We create a hipSharedMemConfig type variable to store in it the configuration of 8-byte mode for shared memory in the GPU.
	hipDeviceSetSharedMemConfig(pConfig); // We set the 8-byte mode for shared memory in the selected GPU.
	
	// We create the pointers to the data that the selected GPU will require.
	int mPlusOne = neuron->m + 1; // This value is repetitively used and strategically stored here for performance purposes.
	double *d_X; // This pointer variable is used to store the data from "neuron->X" into the selected GPU.
	double *d_Y; // This pointer variable is used to store the data from "neuron->Y" into the selected GPU.
	double *d_w_new; // This pointer variable is used to store the data from "neuron->w_new" into the selected GPU.
	double *d_TransposeOf_X_tilde; // This pointer variable is used to store the transpose of the transformed version of "d_X" into "X_tilde" in the selected GPU.
	double *d_f_x_tilde; // This pointer variable is used to store the output of the body of the neuron in the selected GPU.
	double *d_A_u; // This pointer variable is used to store the output of the application of the chosen activation function in the selected GPU.
	double *d_dA_u; // This pointer variable is used to store the output of the application of the derivative of the chosen activation function in the selected GPU.
	double *d_accuracyTerm1; // This pointer variable is used to store key data that is required to calculate the adjusted R squared of the model generated by the neuron.
	double *d_accuracyTerm2; // This pointer variable is used to store key data that is required to calculate the adjusted R squared of the model generated by the neuron.
	double *d_reducedAccuracyTerm1; // This pointer variable is used to store the data of "d_accuracyTerm1", but after having applied the Parallel Reduction Strategy.
	double *d_reducedAccuracyTerm2; // This pointer variable is used to store the data of "d_accuracyTerm2", but after having applied the Parallel Reduction Strategy.
	double *d_errorTerm; // This pointer variable is used to store key data for the calculation of the error term that is applied in the learning process of the artificial neuron.
	double *d_errorTerm_dot_Xtilde; // This pointer variable is used to store the data of "d_errorTerm", but after having applied the Parallel Reduction Strategy.
	
	// We allocate the required memory in the selected GPU.
	CHECK(hipMalloc((void **) &d_X, neuron->n*neuron->m*sizeof(double)));
	CHECK(hipMalloc((void **) &d_Y, neuron->n*sizeof(double)));
	int w_new_Bytes = mPlusOne*sizeof(double); // This variable stores the number of bytes to allocate the new weight values that will be obtained each epoch of the training process, for performance purposes.
	CHECK(hipMalloc((void **) &d_w_new, w_new_Bytes));
	CHECK(hipMalloc((void **) &d_TransposeOf_X_tilde, mPlusOne*neuron->n*sizeof(double)));
	CHECK(hipMalloc((void **) &d_f_x_tilde, neuron->n*sizeof(double)));
	CHECK(hipMalloc((void **) &d_A_u, neuron->n*sizeof(double)));
	CHECK(hipMalloc((void **) &d_dA_u, neuron->n*sizeof(double)));
	CHECK(hipMalloc((void **) &d_accuracyTerm1, neuron->n*sizeof(double)));
	CHECK(hipMalloc((void **) &d_accuracyTerm2, neuron->n*sizeof(double)));
	int parRed_Bytes = trueUnrollingSize1*sizeof(double); // This variable stores the number of bytes to allocate in those variables that will have the Unrolling Parallel Reduction strategy applied once, for performance purposes.
	CHECK(hipMalloc((void **) &d_reducedAccuracyTerm1, parRed_Bytes));
	CHECK(hipMalloc((void **) &d_reducedAccuracyTerm2, parRed_Bytes));
	CHECK(hipMalloc((void **) &d_errorTerm, mPlusOne*neuron->n*sizeof(double)));
	int errorTerm_dot_Xtilde_Bytes = mPlusOne*trueUnrollingSize1*sizeof(double); // This variable stores the number of bytes to allocate the device variable "d_errorTerm_dot_Xtilde", for performance purposes.
	CHECK(hipMalloc((void **) &d_errorTerm_dot_Xtilde, errorTerm_dot_Xtilde_Bytes));
	
	
	// --------------- PREPROCESSING OF THE INPUT DATA --------------- //
	// We transfer the input data that the neuron will need into the selected GPU.
	CHECK(hipMemcpy(d_X, neuron->X, (neuron->n*neuron->m*sizeof(double)), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_Y, neuron->Y, (neuron->n*sizeof(double)), hipMemcpyHostToDevice));
	
	// We obtain the transpose of "X_tilde" in the GPU.
	getTransposeOfInputData_singleGPU <<< grid_n, block_32x_1y >>> (d_X, neuron->n, mPlusOne, d_TransposeOf_X_tilde);
	
	
	// -------------------- WEIGHT INITIALIZATION -------------------- //
	// Store the initial weight values into "neuron->w_new" and into "neuron->w_best" sequentially.
	if (neuron->isInitial_w == 0) {
		// In order to initialize "neuron->w_new" with random values between "-1" and "+1"., intialize random number generator.
		time_t t;
		srand((unsigned) time(&t));
	    
		// Initialize "neuron->w_new" with random values between -1 to +1 with three decimals at the most. Give the save values to "neuron->w_best".
		double currentRandomNumber;
		for (int current_w=0 ; current_w<mPlusOne; current_w++) {
			currentRandomNumber = ((float) (rand() % 1000))/500 - 1;
			neuron->w_first[current_w] = currentRandomNumber;
			neuron->w_new[current_w] = currentRandomNumber;
			neuron->w_best[current_w] = currentRandomNumber;
		}
	} else if (neuron->isInitial_w == 1) {
		// Pass the values of "neuron->w_first" to "neuron->w_new" and "neuron->w_best".
		for (int current_w=0 ; current_w<mPlusOne; current_w++) {
			neuron->w_new[current_w] = neuron->w_first[current_w];
			neuron->w_best[current_w] = neuron->w_first[current_w];
		}
	}
	
	// We pass the generated weights to the GPU.
	CHECK(hipMemcpy(d_w_new, neuron->w_new, w_new_Bytes, hipMemcpyHostToDevice));
	
	// We allocate all the memory that will be required in the CPU for the training process of the neuron.
	int double32_Bytes = 32*sizeof(double); // This variable stores the number of bytes required to store 32 double variable type, for performance purposes.
	int double64_Bytes = 2*double32_Bytes; // This variable stores the number of bytes required to store 64 double variable type, for performance purposes.
	int double96_Bytes = 3*double32_Bytes; // This variable stores the number of bytes required to store 96 double variable type, for performance purposes.
	int double128_Bytes = 2*double64_Bytes; // This variable stores the number of bytes required to store 128 double variable type, for performance purposes.
	int unrollingTotalBlockSize1 = grid_n.x * block_32x_1y.x; // This variable stores the total number of GPU threads that are to be employed specifically for calling a GPU Kernel to apply the Parallel Reduction strategy to a certain set of data. NOTE: This variable has more application/sense when applying the Parallel Reduction two or more consecutive times.
	double *h_reducedAccuracyTerm1 = (double *) malloc(parRed_Bytes); // CPU Allocated variable that will contain all the individual contributions made by each thread block in an attemp to apply the parallel reduction strategy to "d_accuracyTerm1".
	double *h_reducedAccuracyTerm2 = (double *) malloc(parRed_Bytes); // CPU Allocated variable that will contain all the individual contributions made by each thread block in an attemp to apply the parallel reduction strategy to "d_accuracyTerm2".
	double totalSumOfAccuracyTerm1 = 0; // This variable is used to sequentially sum all the contributions of each GPU block that were made to get "d_accuracyTerm1" and that were stored in "h_reducedAccuracyTerm1".
	double totalSumOfAccuracyTerm2 = 0; // This variable is used to sequentially sum all the contributions of each GPU block that were made to get "d_accuracyTerm2" and that were stored in "h_reducedAccuracyTerm2".
	int nMinusOne = neuron->n-1; // This variable is used to store a repetitive value that is used several times in the program, for performance purposes.
	double currentAccuracy = 0; // This variable is used to contain the current accuracy of the neuron.
	double *idata; // This variable is used to convert a pointer of interest to have a new origin from such pointer.
	double *h_errorTerm_dot_Xtilde = (double *) malloc(errorTerm_dot_Xtilde_Bytes); // CPU allocated variable that will contain all the individual contributions made by each thread block in an attemp to apply the parallel reduction strategy to "d_errorTerm".
	double totalErrorTerm_dot_Xtilde = 0; // This variable is used to sum all the contributions of each GPU block that were made to get "d_errorTerm" and that were stored in "d_errorTerm_dot_Xtilde".
	double *w_old = (double *) malloc(w_new_Bytes); // Allocate the memory required for the variable "w_old", which will contain the previous weight values that were obtained with respect to the current ones.
	
	
	// ------------------------------------- //
	// ----- REGRESSION MODEL SELECTED ----- //
	// ------------------------------------- //
	
	// ----------- EVALUATION OF THE INITIAL WEIGHT VALUES ----------- //
	// We calculate "f_x_tilde", "A(u)", "dA(u)" and "the part 1 of the accuracy terms".
	getFxTilde_Au_dAu_and_accuracyTermsPart1_singleGPU <<< grid_n, block_32x_1y, double96_Bytes >>> (d_X, d_Y, d_w_new, neuron->n, neuron->m, neuron->activationFunctionToBeUsed, d_f_x_tilde, d_A_u, d_dA_u, d_accuracyTerm1, d_accuracyTerm2);
	CHECK(hipDeviceSynchronize()); // We force the program to wait until all GPU threads have finished the last task they were given.
	getParallelReduction <<< trueUnrollingSize1, block_32x_1y, double32_Bytes >>> (d_accuracyTerm1, d_reducedAccuracyTerm1, unrollingTotalBlockSize1, numberOfUnrollingLoop1); // We apply the parallel reduction strategy on "d_accuracyTerm1".
	getParallelReduction <<< trueUnrollingSize1, block_32x_1y, double32_Bytes >>> (d_accuracyTerm2, d_reducedAccuracyTerm2, unrollingTotalBlockSize1, numberOfUnrollingLoop1); // We apply the parallel reduction strategy on "d_accuracyTerm2".
	
	// We calculate the sequential part of "the part 1 of the accuracy terms" by sequentially summing all the contributions made and stored in "d_reducedAccuracyTerm1" and "d_reducedAccuracyTerm2" after having applied the parallel reduction strategy on them.
	CHECK(hipMemcpy(h_reducedAccuracyTerm1, d_reducedAccuracyTerm1, parRed_Bytes, hipMemcpyDeviceToHost)); // We transfer the GPU data from "d_reducedAccuracyTerm1" to the CPU through "h_reducedAccuracyTerm1".
	CHECK(hipMemcpy(h_reducedAccuracyTerm2, d_reducedAccuracyTerm2, parRed_Bytes, hipMemcpyDeviceToHost)); // We transfer the GPU data from "d_reducedAccuracyTerm2" to the CPU through "h_reducedAccuracyTerm2".
	totalSumOfAccuracyTerm1 = 0; // We reset the value of the accuracy term 1, in which we will store the value of SSE.
	totalSumOfAccuracyTerm2 = 0; // We reset the value of the accuracy term 2, in which we will temporarily store the sum of all the values from the "real output matrix".
	for (int currentBlock=0; currentBlock<trueUnrollingSize1; currentBlock++) {
		totalSumOfAccuracyTerm1 += h_reducedAccuracyTerm1[currentBlock]; // We sum all the fragments of the SSE that was calculated by the previous parallelization process.
		totalSumOfAccuracyTerm2 += h_reducedAccuracyTerm2[currentBlock]; // We sum all the fragments of the "real output matrix sum" that was calculated by the previous parallelization process.
	}
	h_reducedAccuracyTerm2[0] = totalSumOfAccuracyTerm2 / neuron->n; // We calculate the mean of the values contained in the "real output matrix".
	
	// We calculate "the part 2 of the accuracy terms".
	CHECK(hipMemcpy(d_reducedAccuracyTerm2, h_reducedAccuracyTerm2, sizeof(double), hipMemcpyHostToDevice)); // We pass mean of the "real output matrix" to the GPU, which is contained in the first data location of the pointer variable "h_reducedAccuracyTerm2".
	getNeuronAdjustedCoefficientOfDetermination_singleGPUvoidPart2 <<< grid_n, block_32x_1y, double64_Bytes >>> (d_Y, neuron->n, d_accuracyTerm1, d_reducedAccuracyTerm2);
	CHECK(hipDeviceSynchronize()); // We force the program to wait until all GPU threads have finished the last task they were given.
	getParallelReduction <<< trueUnrollingSize1, block_32x_1y, double32_Bytes >>> (d_accuracyTerm1, d_reducedAccuracyTerm1, unrollingTotalBlockSize1, numberOfUnrollingLoop1); // We apply the parallel reduction strategy on "d_accuracyTerm1", containing the SST data.
	CHECK(hipMemcpy(h_reducedAccuracyTerm1, d_reducedAccuracyTerm1, parRed_Bytes, hipMemcpyDeviceToHost)); // We transfer the GPU data from "d_reducedAccuracyTerm1" to the CPU through "h_reducedAccuracyTerm1".
	totalSumOfAccuracyTerm2 = 0; // We reset the value of the accuracy term 2, in which we will store the value of SST.
	for (int currentBlock=0; currentBlock<trueUnrollingSize1; currentBlock++) {
		totalSumOfAccuracyTerm2 += h_reducedAccuracyTerm1[currentBlock]; // We sum all the fragments of the SST that was calculated by the previous parallelization process.
	}

	// Finally, we calculate the adjusted coefficient of determination and store its results in the variable "currentAccuracy".
	currentAccuracy = 1 - ( (totalSumOfAccuracyTerm1/(nMinusOne-(neuron->m)))/(totalSumOfAccuracyTerm2 / nMinusOne) );
	
	// We pass the current accuracy to the best accuracy record because this is the evaluation of the very first weight values.
	neuron->bestAccuracy = currentAccuracy;
	
	// If the desired accuracy has been reached, then conclude the training process of the neuron. Otherwise, continue training it.
	if (currentAccuracy > neuron->stopAboveThisAccuracy) {
		printf("\nThe adjusted R squared (%f) of the neuron has achieved a higher one with respect to the one that was specified as a goal the very first instant it was created.\n", currentAccuracy);
		
		// Before terminating this function, we free the GPU and CPU allocated memory since they will no longer be used.
		CHECK(hipFree(d_X));
		CHECK(hipFree(d_Y));
		CHECK(hipFree(d_w_new));
		CHECK(hipFree(d_TransposeOf_X_tilde));
		CHECK(hipFree(d_f_x_tilde));
		CHECK(hipFree(d_A_u));
		CHECK(hipFree(d_dA_u));
		CHECK(hipFree(d_accuracyTerm1));
		CHECK(hipFree(d_accuracyTerm2));
		CHECK(hipFree(d_reducedAccuracyTerm1));
		CHECK(hipFree(d_reducedAccuracyTerm2));
		CHECK(hipFree(d_errorTerm));
		CHECK(hipFree(d_errorTerm_dot_Xtilde));
		free(h_reducedAccuracyTerm1);
		free(h_reducedAccuracyTerm2);
		free(h_errorTerm_dot_Xtilde);
		free(w_old);
		return;
	}
	
	// -------- BEGINNING OF THE EPOCHS OF THE MODEL ------- //
	for (int currentEpoch=0; currentEpoch<(neuron->maxEpochs); currentEpoch++) {
		// Pass the data of "neuron->w_new" to "w_old".
		for (int currentCoefficient=0; currentCoefficient<mPlusOne; currentCoefficient++) {
			w_old[currentCoefficient] = neuron->w_new[currentCoefficient];
		}
		
		// Calculate the error term obtainable with the current weight values so that we can later update the current weight values ("w_old") in order to obtain the new ones ("neuron->w_new").
		getErrorAndUpdateWeightValues_singleGPUpart1 <<< grid_n, block_32x_1y, double128_Bytes >>> (d_TransposeOf_X_tilde, d_Y, neuron->n, mPlusOne, d_A_u, d_dA_u, d_errorTerm);
		CHECK(hipDeviceSynchronize()); // We force the program to wait until all GPU threads have finished the last task they were given.
		getErrorAndUpdateWeightValues_singleGPUpart2 <<< trueUnrollingSize1, block_32x_1y, double32_Bytes >>> (d_errorTerm, neuron->n, mPlusOne, trueUnrollingSize1, unrollingTotalBlockSize1, numberOfUnrollingLoop1, d_errorTerm_dot_Xtilde);
		CHECK(hipMemcpy(h_errorTerm_dot_Xtilde, d_errorTerm_dot_Xtilde, errorTerm_dot_Xtilde_Bytes, hipMemcpyDeviceToHost)); // We transfer the GPU data from "d_errorTerm_dot_Xtilde" to the CPU through "h_errorTerm_dot_Xtilde".
		
		// We update the current weight values ("w_old") in order to obtain the new ones ("neuron->w_new") by sequentially summing all the individual contributions made after having applied the parallel reduction strategy on "d_errorTerm", whose result was stored in "h_errorTerm_dot_Xtilde".
		idata = h_errorTerm_dot_Xtilde; // We convert the pointer of interest from "h_errorTerm_dot_Xtilde" to be the origin pointer of "idata".
		for (int currentRow=0; currentRow<mPlusOne; currentRow++) {
			totalErrorTerm_dot_Xtilde = 0; // We reset the value of "totalErrorTerm_dot_Xtilde" to sum the contributed error values for the next weight.
			for (int currentBlock=0; currentBlock<trueUnrollingSize1; currentBlock++) {
				totalErrorTerm_dot_Xtilde += idata[currentBlock];
			}
			neuron->w_new[currentRow] = w_old[currentRow] + neuron->learningRate * totalErrorTerm_dot_Xtilde; // We update the current weight value.
			idata += trueUnrollingSize1; // We mode the pointer of "h_errorTerm_dot_Xtilde" to the next row/weight.
		}
		CHECK(hipMemcpy(d_w_new, neuron->w_new, w_new_Bytes, hipMemcpyHostToDevice)); // We pass the values of "neuron->w_new" to the GPU, through its pointer variable "d_w_new".
		
		// We recalculate "f_x_tilde", "A(u)", "dA(u)" and "the part 1 of the accuracy terms".
		getFxTilde_Au_dAu_and_accuracyTermsPart1_singleGPU <<< grid_n, block_32x_1y, double96_Bytes >>> (d_X, d_Y, d_w_new, neuron->n, neuron->m, neuron->activationFunctionToBeUsed, d_f_x_tilde, d_A_u, d_dA_u, d_accuracyTerm1, d_accuracyTerm2);
		CHECK(hipDeviceSynchronize()); // We force the program to wait until all GPU threads have finished the last task they were given.
		getParallelReduction <<< trueUnrollingSize1, block_32x_1y, double32_Bytes >>> (d_accuracyTerm1, d_reducedAccuracyTerm1, unrollingTotalBlockSize1, numberOfUnrollingLoop1); // We apply the parallel reduction strategy on "d_accuracyTerm1".
		getParallelReduction <<< trueUnrollingSize1, block_32x_1y, double32_Bytes >>> (d_accuracyTerm2, d_reducedAccuracyTerm2, unrollingTotalBlockSize1, numberOfUnrollingLoop1); // We apply the parallel reduction strategy on "d_accuracyTerm2".
		
		// We recalculate the sequential part of "the part 1 of the accuracy terms" by sequentially summing all the contributions made and stored in "d_reducedAccuracyTerm1" and "d_reducedAccuracyTerm2" after having applied the parallel reduction strategy on them.
		CHECK(hipMemcpy(h_reducedAccuracyTerm1, d_reducedAccuracyTerm1, parRed_Bytes, hipMemcpyDeviceToHost)); // We transfer the GPU data from "d_reducedAccuracyTerm1" to the CPU through "h_reducedAccuracyTerm1".
		CHECK(hipMemcpy(h_reducedAccuracyTerm2, d_reducedAccuracyTerm2, parRed_Bytes, hipMemcpyDeviceToHost)); // We transfer the GPU data from "d_reducedAccuracyTerm2" to the CPU through "h_reducedAccuracyTerm2".
		totalSumOfAccuracyTerm1 = 0; // We reset the value of the accuracy term 1, in which we will store the value of SSE.
		totalSumOfAccuracyTerm2 = 0; // We reset the value of the accuracy term 2, in which we will temporarily store the sum of all the values from the "real output matrix".
		for (int currentBlock=0; currentBlock<trueUnrollingSize1; currentBlock++) {
			totalSumOfAccuracyTerm1 += h_reducedAccuracyTerm1[currentBlock]; // We sum all the fragments of the SSE that was calculated by the previous parallelization process.
			totalSumOfAccuracyTerm2 += h_reducedAccuracyTerm2[currentBlock]; // We sum all the fragments of the "real output matrix sum" that was calculated by the previous parallelization process.
		}
		h_reducedAccuracyTerm2[0] = totalSumOfAccuracyTerm2 / neuron->n; // We calculate the mean of the values contained in the "real output matrix".
		
		// We recalculate "the part 2 of the accuracy terms".
		CHECK(hipMemcpy(d_reducedAccuracyTerm2, h_reducedAccuracyTerm2, sizeof(double), hipMemcpyHostToDevice)); // We pass mean of the "real output matrix" to the GPU, which is contained in the first data location of the pointer variable "h_reducedAccuracyTerm2".
		getNeuronAdjustedCoefficientOfDetermination_singleGPUvoidPart2 <<< grid_n, block_32x_1y, double64_Bytes >>> (d_Y, neuron->n, d_accuracyTerm1, d_reducedAccuracyTerm2);
		CHECK(hipDeviceSynchronize()); // We force the program to wait until all GPU threads have finished the last task they were given.
		getParallelReduction <<< trueUnrollingSize1, block_32x_1y, double32_Bytes >>> (d_accuracyTerm1, d_reducedAccuracyTerm1, unrollingTotalBlockSize1, numberOfUnrollingLoop1); // We apply the parallel reduction strategy on "d_accuracyTerm1", containing the SST data.
		CHECK(hipMemcpy(h_reducedAccuracyTerm1, d_reducedAccuracyTerm1, parRed_Bytes, hipMemcpyDeviceToHost)); // We transfer the GPU data from "d_reducedAccuracyTerm1" to the CPU through "h_reducedAccuracyTerm1".
		totalSumOfAccuracyTerm2 = 0; // We reset the value of the accuracy term 2, in which we will store the value of SST.
		for (int currentBlock=0; currentBlock<trueUnrollingSize1; currentBlock++) {
			totalSumOfAccuracyTerm2 += h_reducedAccuracyTerm1[currentBlock]; // We sum all the fragments of the SST that was calculated by the previous parallelization process.
		}

		// Finally, we recalculate the adjusted coefficient of determination and store its results in the variable "currentAccuracy".
		currentAccuracy = 1 - ( (totalSumOfAccuracyTerm1/(nMinusOne-(neuron->m)))/(totalSumOfAccuracyTerm2 / nMinusOne) );
		
		// We compare the accuracy of the currently obtained weight values with respect to the latest best one recorded. If the current one is better than the recorded one, then store the current one in its place and do the same for the best recorded weight values.
		if ((currentAccuracy) > (neuron->bestAccuracy)) {
			neuron->bestAccuracy = currentAccuracy; // Pass the value of the current accuracy into "neuron->bestAccuracy".
			for (int current_w=0 ; current_w<mPlusOne; current_w++) { // Pass the values of "neuron->w_new" to "neuron->w_best".
			neuron->w_best[current_w] = neuron->w_new[current_w];
		    }
		}
		
		// Determine whether it was requested that the neuron reports its learning progress or not.
		if (neuron->isReportLearningProgress == 1) { // If the implementer requested the neuron to report its progress, apply the following code.
			if ((currentEpoch % neuron->reportEachSpecifiedEpochs) == 0) { // Make neuron report at each "neuron->reportEachSpecifiedEpochs" epochs.
			    printf("\nEpoch %d --> single neuron in DNN has achieved an adjusted R squared of %f\n", currentEpoch+1, currentAccuracy);
			}
		}
		
		// If the desired accuracy has been reached, then conclude the training process of the neuron. Otherwise, continue training it.
		if (currentAccuracy > neuron->stopAboveThisAccuracy) {
			printf("\nThe adjusted R squared (%f) of the neuron has achieved a higher one with respect to the one that was specified as a goal when concluding the epoch number %d.\n", currentAccuracy, currentEpoch+1);
			
			// Before terminating this function, we free the GPU and CPU allocated memory since they will no longer be used.
			CHECK(hipFree(d_X));
			CHECK(hipFree(d_Y));
			CHECK(hipFree(d_w_new));
			CHECK(hipFree(d_TransposeOf_X_tilde));
			CHECK(hipFree(d_f_x_tilde));
			CHECK(hipFree(d_A_u));
			CHECK(hipFree(d_dA_u));
			CHECK(hipFree(d_accuracyTerm1));
			CHECK(hipFree(d_accuracyTerm2));
			CHECK(hipFree(d_reducedAccuracyTerm1));
			CHECK(hipFree(d_reducedAccuracyTerm2));
			CHECK(hipFree(d_errorTerm));
			CHECK(hipFree(d_errorTerm_dot_Xtilde));
			free(h_reducedAccuracyTerm1);
			free(h_reducedAccuracyTerm2);
			free(h_errorTerm_dot_Xtilde);
			free(w_old);
			return;
		}
	}
	
	// Determine whether it was requested that the neuron reports its learning progress or not.
	if (neuron->isReportLearningProgress == 1) { // If the implementer requested the neuron to report its progress, apply the following code.
		// Make the neuron report its last progress made.
		printf("\nEpoch %d --> single neuron in DNN has achieved an adjusted R squared of %f\n", neuron->maxEpochs, currentAccuracy);
	}
	
	// Before terminating this function, we free the GPU and CPU allocated memory since they will no longer be used.
	CHECK(hipFree(d_X));
	CHECK(hipFree(d_Y));
	CHECK(hipFree(d_w_new));
	CHECK(hipFree(d_TransposeOf_X_tilde));
	CHECK(hipFree(d_f_x_tilde));
	CHECK(hipFree(d_A_u));
	CHECK(hipFree(d_dA_u));
	CHECK(hipFree(d_accuracyTerm1));
	CHECK(hipFree(d_accuracyTerm2));
	CHECK(hipFree(d_reducedAccuracyTerm1));
	CHECK(hipFree(d_reducedAccuracyTerm2));
	CHECK(hipFree(d_errorTerm));
	CHECK(hipFree(d_errorTerm_dot_Xtilde));
	free(h_reducedAccuracyTerm1);
	free(h_reducedAccuracyTerm2);
	free(h_errorTerm_dot_Xtilde);
	free(w_old);
	
	printf("\nThe best adjusted R squared (%f) achieved by the neuron did not surpased the defined goal but its training process has been successfully concluded.\n", neuron->bestAccuracy);
	return;
}


/**
* The "getTransposeOfInputData_singleGPU()" global static function is used to
* apply a single GPU to calculate and store the transpose of the input matrix in
* its transformed form of "X_tilde", which will be used to train a single
* artificial neuron.
* 
* 
* @param double *X - This argument will contain the pointer to a memory
*		allocated input matrix, from which the desired machine learning
*		algorithm will be calculated. THIS VARIABLE SHOULD BE ALLOCATED
* 		AND INITIALIZED BEFORE CALLING THIS FUNCTION WITH A SIZE OF "n"
* 		TIMES "m" 'DOUBLE' MEMORY SPACES.
*
* @param int n - This argument will represent the total number of samples (rows)
* 		that the input matrix has, with which the output data was
*		obtained.
*
* @param int mPlusOne - This argument will represent the total number of
*		features (independent variables) that the input matrix has plus
* 		one.
*
* @param double *TransposeOf_X_tilde - This argument will contain the pointer to
* 		a memory allocated matrix in which the transpose of the argument
* 		variable "X" in its transformed form of "X_tilde" will be stored.
* 		THIS VARIABLE SHOULD BE ALLOCATED BEFORE CALLING THIS FUNCTION
* 		WITH A SIZE OF "n" TIMES "m+1" TIMES "n" 'DOUBLE' MEMORY SPACES.
* 
*
* NOTE: RESULTS ARE STORED IN "TransposeOf_X_tilde".
* 
* @return void
*
* @author Miranda Meza Cesar
* CREATION DATE: JANUARY 21, 2022
* LAST UPDATE: N/A
*/
__global__ static void getTransposeOfInputData_singleGPU(double *X, int n, int mPlusOne, double *TransposeOf_X_tilde) {
	// We obtain the GPU thread global coordinate.
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	
	// We calculate the transpose of "X_tilde" by using the argument variable "X", but only if it is within the threads boundary.
	if (idx < n) {
		double *idata = X; // We convert the pointer of interest from "X" to be the origin pointer of "idata".
		double *odata = TransposeOf_X_tilde; // We convert the pointer of interest from "TransposeOf_X_tilde" to be the origin pointer of "odata".
		odata[idx] = 1;
		for (int currentColumn=1; currentColumn<(mPlusOne); currentColumn++) {
			odata += n; // We move the origin pointer of the argument variable "TransposeOf_X_tilde" to its next column.
			odata[idx] = idata[idx]; // We apply the transpose with respect to the next column of "X_tilde".
			idata += n; // We move the origin pointer of the argument varaible "X" to its next row.
		}
	}
	
	return;
}


/**
* The "getFxTilde_Au_dAu_and_accuracyTermsPart1_singleGPU()" global static
* function is used to apply a single GPU to calculate "f(\tilde{x})", A(u),
* dA(u) and the first part of the accuracy terms calculations.
* 
* @param double *X - This argument will contain the pointer to a memory
*		allocated input matrix, from which the desired machine learning
*		algorithm will be calculated. THIS VARIABLE SHOULD BE ALLOCATED
* 		AND INITIALIZED BEFORE CALLING THIS FUNCTION WITH A SIZE OF "n"
* 		TIMES "m" 'DOUBLE' MEMORY SPACES.
*
* @param double *Y - This argument will contain the pointer to a memory
* 		allocated output matrix, representing the real data of the
*		system under study. THIS VARIABLE SHOULD BE ALLOCATED AND
* 		INITIALIZED BEFORE CALLING THIS FUNCTION WITH A SIZE OF "n"
* 		TIMES "p=1" 'DOUBLE' MEMORY SPACES.
* 
* @param double *w_new - This argument will contain the pointer to a memory
*			allocated variable in which we will store the last
*			identified coefficient values for the model of a single
* 			neuron in Deep Neural Network. These coefficients will
* 			each be stored in the same row but under different
* 			columns where the first coefficient (b_0) will be stored
* 			in the column with index 0; the second coefficient (w_1)
* 			will be stored in the column index 1 and; the last
* 			coefficient (w_m) will be stored in the column index m.
* 			IT IS INDISPENSABLE THAT THIS VARIABLE IS ALLOCATED AND
* 			INITIALIZED BEFORE CALLING THIS FUNCTION WITH A VARIABLE
* 			SIZE OF "1" TIMES "m+1" 'DOUBLE' MEMORY SPACES.
* 
* @param int n - This argument will represent the total number of samples (rows)
* 		that the input matrix has, with which the output data was
*		obtained.
*
* @param int m - This argument will represent the total number of features
* 		(independent variables) that the input matrix has, with which
* 		the output data was obtained.
*
* @param int activationFunctionToBeUsed - This argument will represent the
* 					identifier of the desired activation
* 					function to be used by the neuron during
* 					its training process. Its possible valid
* 					values are the following:
*					0 = Rectified Linear Units (ReLU).
*					1 = Hyperbolic tangent (tanh).
*					2 = Logistic function.
*					3 = Raise to the 1st power.
*					4 = Raise to the 2nd power.
*					5 = Raise to the 3rd power.
*					6 = Raise to the 4th power.
*					7 = Raise to the 5th power.
*					8 = Raise to the 6th power.
*					9 = 1st order degree exponential.
*					10 = 2nd order degree exponential.
*
* @param double *f_x_tilde - This argument will contain the pointer to a memory
* 			allocated matrix that is used to store the output of the
* 			body of the neuron in the selected GPU. IT IS
* 			INDISPENSABLE THAT THIS VARIABLE IS ALLOCATED BEFORE
* 			CALLING THIS FUNCTION WITH A VARIABLE SIZE OF "n" TIMES
* 			"1" 'DOUBLE' MEMORY SPACES.
*
* @param double *A_u - This argument will contain the pointer to a memory
* 		allocated output matrix in which the requested activation
* 		function will be applied on the argument pointer variable
* 		"f_x_tilde" and its result will be saved in "A_u". "A_u" SHOULD
* 		BE ALLOCATED BEFORE CALLING THIS FUNCTION WITH A SIZE OF "n"
* 		TIMES "p=1" 'DOUBLE' MEMORY SPACES.
*
* @param double *dA_u - This argument will contain the pointer to a memory
* 		allocated output matrix in which the derivate of the activation
* 		function will be applied and its result will be saved in "dA_u".
* 		"dA_u" SHOULD BE ALLOCATED BEFORE CALLING THIS FUNCTION WITH A
* 		SIZE OF "n" TIMES "p=1" 'DOUBLE' MEMORY SPACES.
*
* @param double *accuracyTerm1 - This argument will contain the pointer to a
* 			memory allocated matrix that will contain all the
* 			calculations required to obtained the SSE value (with the
* 			intention of applying the Parallel Reduction strategy to
* 			it on another process, external to this function). IT IS
* 			INDISPENSABLE THAT THIS VARIABLE IS ALLOCATED BEFORE
* 			CALLING THIS FUNCTION WITH A VARIABLE SIZE OF "1" TIMES
* 			"n" 'DOUBLE' MEMORY SPACES.
*
* @param double *accuracyTerm2 - This argument will contain the pointer to a
* 			memory allocated matrix that will contain all the
* 			calculations required to obtained the sum of all the
* 			values contained in the argument pointer variable "Y"
* 			(with the intention of applying the Parallel Reduction
* 			strategy to it on another process, external to this
* 			function). IT IS INDISPENSABLE THAT THIS VARIABLE IS
* 			ALLOCATED BEFORE CALLING THIS FUNCTION WITH A VARIABLE
* 			SIZE OF "1" TIMES "n" 'DOUBLE' MEMORY SPACES.
* 
*
* NOTE: RESULTS ARE STORED IN "f_x_tilde", "A_u", "dA_u", "accuracyTerm1" AND
*	"accuracyTerm2".
* 
* @return void
*
* @author Miranda Meza Cesar
* CREATION DATE: JANUARY 23, 2022
* LAST UPDATE: JANUARY 25, 2022
*/
__global__ static void getFxTilde_Au_dAu_and_accuracyTermsPart1_singleGPU(double *X, double *Y, double *w_new, int n, int m, int activationFunctionToBeUsed, double *f_x_tilde, double *A_u, double *dA_u, double *accuracyTerm1, double *accuracyTerm2) {
	// We obtain the GPU thread coordinates.
	int idx = threadIdx.x + blockIdx.x * blockDim.x; // We obtain the GPU thread global coordinate.
	int tid = threadIdx.x; // We obtain the GPU thread local coordinate
	
	// If the current GPU thread is within boundary, then proceed to work with the task. Otherwise, conclude your operation.
	if (idx < n) {
		// We calculate the values of "f(x_tilde)".
		getFxTilde(X, w_new, m, f_x_tilde, tid, idx);
		
		// We calculate the currently predicted output data made by the neuron and store it in "A_u" by applying the desired activation function to "f_x_tilde".
		getActivationFunction(activationFunctionToBeUsed, f_x_tilde, A_u, idx); // We calculate A(u) and store it in the pointer variable "A_u".
		
		// We calculate the derivative of A(u).
		// NOTE: Remember that "Y_hat" = A(u) = "A_u".
		getDerivateOfActivationFunction(activationFunctionToBeUsed, f_x_tilde, A_u, dA_u, idx); // We calculate the derivative of A(u) and store it in the pointer variable "dA_u".
		
		// We calculate the part 1 of the corresponding evaluation metric with respect to the actual data of the system under study "Y" and the currently predicted output made by the neuron "A_u".
		getNeuronAdjustedCoefficientOfDetermination_singleGPUPart1(Y, A_u, accuracyTerm1, accuracyTerm2, idx); // We calculate the part 1 of the calculation of the current adjusted coefficient of determination of the neuron.
	}
	
	return;
}
/**
* The "getFxTilde()" device static function is used to apply a single
* GPU to calculate and store the f(x_tilde) value, which stands for the output
* of the body of the single neuron in DNN.
*
*
* @param double *X - This argument will contain the pointer to a memory
*		allocated input matrix, from which the desired machine learning
*		algorithm will be calculated. THIS VARIABLE SHOULD BE ALLOCATED
* 		AND INITIALIZED BEFORE CALLING THIS FUNCTION WITH A SIZE OF "n"
* 		TIMES "m" 'DOUBLE' MEMORY SPACES.
*
* @param double *Y - This argument will contain the pointer to a memory
* 		allocated output matrix, representing the real data of the
*		system under study. THIS VARIABLE SHOULD BE ALLOCATED AND
* 		INITIALIZED BEFORE CALLING THIS FUNCTION WITH A SIZE OF "n"
* 		TIMES "p=1" 'DOUBLE' MEMORY SPACES.
* 
* @param double *w_new - This argument will contain the pointer to a memory
*			allocated variable in which we will store the last
*			identified coefficient values for the model of a single
* 			neuron in Deep Neural Network. These coefficients will
* 			each be stored in the same row but under different
* 			columns where the first coefficient (b_0) will be stored
* 			in the column with index 0; the second coefficient (w_1)
* 			will be stored in the column index 1 and; the last
* 			coefficient (w_m) will be stored in the column index m.
* 			IT IS INDISPENSABLE THAT THIS VARIABLE IS ALLOCATED AND
* 			INITIALIZED BEFORE CALLING THIS FUNCTION WITH A VARIABLE
* 			SIZE OF "1" TIMES "m+1" 'DOUBLE' MEMORY SPACES.
* 
* @param int n - This argument will represent the total number of samples (rows)
* 		that the input matrix has, with which the output data was
*		obtained.
*
* @param int m - This argument will represent the total number of features
* 		(independent variables) that the input matrix has, with which
* 		the output data was obtained.
*
* @param int activationFunctionToBeUsed - This argument will represent the
* 					identifier of the desired activation
* 					function to be used by the neuron during
* 					its training process. Its possible valid
* 					values are the following:
*					0 = Rectified Linear Units (ReLU).
*					1 = Hyperbolic tangent (tanh).
*					2 = Logistic function.
*					3 = Raise to the 1st power.
*					4 = Raise to the 2nd power.
*					5 = Raise to the 3rd power.
*					6 = Raise to the 4th power.
*					7 = Raise to the 5th power.
*					8 = Raise to the 6th power.
*					9 = 1st order degree exponential.
*					10 = 2nd order degree exponential.
*
* @param double *f_x_tilde - This argument will contain the pointer to a memory
* 			allocated matrix that is used to store the output of the
* 			body of the neuron in the selected GPU. IT IS
* 			INDISPENSABLE THAT THIS VARIABLE IS ALLOCATED BEFORE
* 			CALLING THIS FUNCTION WITH A VARIABLE SIZE OF "n" TIMES
* 			"1" 'DOUBLE' MEMORY SPACES.
*
* @param int tid - This argument will contain the value of the GPU thread local
* 		coordinate.
*
* @param int idx - This argument will contain the value of the GPU thread global
* 		coordinate.
*
*
* NOTE: RESULT IS STORED IN THE MEMORY ALLOCATED POINTER VARIABLE "f_x_tilde".
* 
* @return void
* 
* @author Miranda Meza Cesar
* CREATION DATE: JANUARY 25, 2022
* LAST UPDATE: N/A
*/
__device__ static void getFxTilde(double *X, double *w_new, int m, double *f_x_tilde, int tid, int idx) {
	// We declare and initialize the shared memory of the GPU that will be used.
	extern __shared__ double sharedMem[]; // We declare the shared memory that we will use for each block.
	// NOTE: Each GPU thread is storing data such that their local address (tid) will represent the identifier of the row number in which they will write data in the shared memory. Moreover, each thread will have assigned 3 columns per row.
	
	// We calculate the values of "f(x_tilde)".
	double *idata_X = X; // We convert the pointer of interest from "X" to be the origin pointer of "idata".
	double *idata_w_new = w_new; // We convert the pointer of interest from "w_new" to be the origin pointer of "idata".
	int tidTimesThree = tid*3; // This variable is used to store a repetitive value that is used several times in the program, for performance purposes.
	int currentRowColumn1 = tidTimesThree; // This variable is used to store a repetitive value that is used several times in the program, for performance purposes.
	int currentRowColumn2 = 1 + currentRowColumn1; // This variable is used to store a repetitive value that is used several times in the program, for performance purposes.
	sharedMem[tidTimesThree] = idata_w_new[0]; // This memory address of the shared memory is where we will store the value of "f(x_tilde)". To begin with such process, we get the bias value of the body of the neuron.
	idata_w_new++; // We move the origin pointer of the argument variable "w_new" to the location of the next weight value, for performance purposes.
	idata_X += idx * m; // We move the origin pointer of the argument variable "X" to the location of the row of interest for the current GPU thread, for performance purposes.
	for (int currentColumn=0; currentColumn<m; currentColumn++) {
		currentRowColumn1++;
		currentRowColumn2++;
		sharedMem[currentRowColumn1] = idata_w_new[currentColumn]; // We pass the current weight value to the shared memory corresponding address.
		sharedMem[currentRowColumn2] = idata_X[currentColumn]; // We pass the current input data to the shared memory corresponding address.
		sharedMem[tidTimesThree] += sharedMem[currentRowColumn1] * sharedMem[currentRowColumn2]; // We multiply the current weight value with the current input data and store it in the address of the shared memory in which we will store the value of "f(x_tilde)".
	}
	f_x_tilde[idx] = sharedMem[tidTimesThree]; // We transfer the result of "f(x_tilde)" from the shared memory to the pointer variable "f_x_tilde".
	
	return;
}


/**
* The "getErrorAndUpdateWeightValues_singleGPUpart1()" global static function is
* used to apply a single GPU to make several calculations required to obtain the
* error term that is applied in the learning process of the artificial neuron
* (with the intention of applying the Parallel Reduction strategy to it on
* another process, external to this function)
* 
*
* @param double *TransposeOf_X_tilde - This argument will contain the pointer to
* 		a memory allocated matrix in which the transpose of the argument
* 		variable "X" in its transformed form of "X_tilde" will be stored.
* 		THIS VARIABLE SHOULD BE ALLOCATED AND INITIALIZED BEFORE CALLING
* 		THIS FUNCTION WITH A SIZE OF "n" TIMES "m+1" TIMES "n" 'DOUBLE'
* 		MEMORY SPACES.
*
* @param double *Y - This argument will contain the pointer to a memory
* 		allocated output matrix, representing the real data of the
*		system under study. THIS VARIABLE SHOULD BE ALLOCATED AND
* 		INITIALIZED BEFORE CALLING THIS FUNCTION WITH A SIZE OF "n"
* 		TIMES "p=1" 'DOUBLE' MEMORY SPACES.
* 
* @param int n - This argument will represent the total number of samples (rows)
* 		that the input matrix has, with which the output data was
*		obtained.
*
* @param int mPlusOne - This argument will represent the total number of
*		features (independent variables) that the input matrix has plus
* 		one.
*
* @param double *A_u - This argument will contain the pointer to a memory
* 		allocated output matrix in which the requested activation
* 		function was applied and stored. "A_u" SHOULD BE ALLOCATED AND
* 		INITIALIZED BEFORE CALLING THIS FUNCTION WITH A SIZE OF "n"
* 		TIMES "p=1" 'DOUBLE' MEMORY SPACES.
*
* @param double *dA_u - This argument will contain the pointer to a memory
* 		allocated output matrix in which the derivate of the activation
* 		function was applied and stored. "dA_u" SHOULD BE ALLOCATED AND
* 		INITIALIZED BEFORE CALLING THIS FUNCTION WITH A SIZE OF "n"
* 		TIMES "p=1" 'DOUBLE' MEMORY SPACES.
*
* @param double *errorTerm - This argument will contain the pointer to a
* 			memory allocated matrix that will contain all the
* 			calculations required to obtained the error term value
* 			that is applied in the learning process of the artificial
* 			neuron (with the intention of applying the Parallel
* 			Reduction strategy to it on another process, external to
* 			this function). IT IS INDISPENSABLE THAT THIS VARIABLE IS
*			ALLOCATED BEFORE CALLING THIS FUNCTION WITH A VARIABLE
* 			SIZE OF "m+1" TIMES "n" 'DOUBLE' MEMORY SPACES.
*
*
* NOTE: RESULT IS STORED IN "errorTerm".
* 
* @return void
*
* @author Miranda Meza Cesar
* CREATION DATE: JANUARY 23, 2022
* LAST UPDATE: N/A
*/
__global__ static void getErrorAndUpdateWeightValues_singleGPUpart1(double *TransposeOf_X_tilde, double *Y, int n, int mPlusOne, double *A_u, double *dA_u, double *errorTerm) {
	// We obtain the GPU thread coordinates.
	int idx = threadIdx.x + blockIdx.x * blockDim.x; // We obtain the GPU thread global coordinate.
	
	// If the current GPU thread is within boundary, then proceed to work with the task. Otherwise, conclude your operation.
	if (idx < n) {
		// We calculate the error term contribution of the current GPU thread.
		double contributedErrorTerm = (Y[idx] - A_u[idx]) * dA_u[idx];
		
		// We calculate the contribution of the current GPU thread to obtain the dot product of the error term and the transposed matrix of X_tilde.
		double *idata1 = TransposeOf_X_tilde; // We convert the pointer of interest from "TransposeOf_X_tilde" to be the origin pointer of "idata".
		double *odata1 = errorTerm; // We convert the pointer of interest from "errorTerm" to be the origin pointer of "odata".
		for (int currentWeight=0; currentWeight<mPlusOne; currentWeight++) {
			odata1[idx] = contributedErrorTerm * idata1[idx]; // We apply the dot product between the error term and the transposed matrix of X_tilde.
			odata1 += n;
			idata1 += n;
		}
	}
	
	return;
}
/**
* The "getErrorAndUpdateWeightValues_singleGPUpart2()" global static function is
* used to employ a single GPU to apply the Parallel Reduction strategy to the
* error term contributions that should have been obtained previously (with the
* function "getErrorAndUpdateWeightValues_singleGPUpart1") to calling this
* function.
* 
*
* @param double *errorTerm - This argument will contain the pointer to a
* 			memory allocated matrix that should contain all the
* 			calculations required to obtained the error term value,
* 			which is obtained through the function
* 			"getErrorAndUpdateWeightValues_singleGPUpart1". As a
* 			result, the function
* 			"getErrorAndUpdateWeightValues_singleGPUpart2" will
* 			apply the Parallel Reduction strategy to "errorTerm",
* 			which should contain all the individual contributions to
* 			obtain the error term having already made a dot product
* 			with "X_tilde". IT IS INDISPENSABLE THAT THIS VARIABLE
* 			IS ALLOCATED AND INITIALIZED BEFORE CALLING THIS
* 			FUNCTION WITH A VARIABLE SIZE OF "m+1" TIMES "n" 'DOUBLE'
* 			MEMORY SPACES.
*
* @param int n - This argument will represent the total number of samples (rows)
* 		that the input matrix has, with which the output data was
*		obtained.
*
* @param int mPlusOne - This argument will represent the total number of
*		features (independent variables) that the input matrix has plus
* 		one.
*
* @param int trueUnrollingSize - This argument is used to represent the true
* 			number of samples that are expected to be obtained after
* 			having applied the Parallel Reduction strategy.
*
* @param int unrollingGridSize - This argument is used to represent the grid
*		size that will be considered for all the GPU Kernels that apply
* 		the Unrolling Parallel Reduction strategy, for performance
* 		purposes.
*
* @param int numberOfUnrollingLoop - This argument is used to represent the
* 				number of unrolling loops that the algorithm will
* 				use when applying the Parallel Reduction
*				Strategy.
*
* @param double *errorTerm_dot_Xtilde - This argument pointer variable is used
*				to store the data of "d_errorTerm", but after
* 				having applied the Parallel Reduction Strategy.
* 				IT IS INDISPENSABLE THAT THIS VARIABLE IS
* 				ALLOCATED BEFORE CALLING THIS FUNCTION WITH A
* 				VARIABLE SIZE OF "m+1" TIMES "GPU grid size"
* 				'DOUBLE' MEMORY SPACES.
*
*
* NOTE: RESULTS ARE STORED IN "errorTerm_dot_Xtilde".
* 
* @return void
*
* @author Miranda Meza Cesar
* CREATION DATE: JANUARY 23, 2022
* LAST UPDATE: N/A
*/
__global__ static void getErrorAndUpdateWeightValues_singleGPUpart2(double *errorTerm, int n, int mPlusOne, int trueUnrollingSize, int unrollingGridSize, int numberOfUnrollingLoop, double *errorTerm_dot_Xtilde) {
	// We apply the parallel reduction strategy to all the individual error term contributions made for each weight available.
	double *idata2 = errorTerm; // We convert the pointer of interest from "errorTerm" to be the origin pointer of "idata".
	double *odata2 = errorTerm_dot_Xtilde; // We convert the pointer of interest from "errorTerm_dot_Xtilde" to be the origin pointer of "odata".
	for (int currentWeight=0; currentWeight<mPlusOne; currentWeight++) {
		getDeviceParallelReduction(idata2, odata2, unrollingGridSize, numberOfUnrollingLoop); // We apply the parallel reduction strategy on "errorTerm".
		idata2 += n;
		odata2 += trueUnrollingSize;
	}
	
	return;
}


/**
* The "getActivationFunction()" device static function is used to apply a single
* GPU to calculate and store the requested activation function to the output of
* the body of a neuron.
*
*
* @param int activationFunctionToBeUsed - This argument will represent the
* 					identifier of the desired activation
* 					function to be used by the neuron during
* 					its training process. Its possible valid
* 					values are the following:
*					0 = Rectified Linear Units (ReLU).
*					1 = Hyperbolic tangent (tanh).
*					2 = Logistic function.
*					3 = Raise to the 1st power.
*					4 = Raise to the 2nd power.
*					5 = Raise to the 3rd power.
*					6 = Raise to the 4th power.
*					7 = Raise to the 5th power.
*					8 = Raise to the 6th power.
*					9 = 1st order degree exponential.
*					10 = 2nd order degree exponential.
*
* @param double *u - This argument will contain the pointer to a memory
*		allocated input matrix, in which the output of the body of a
* 		neuron should be stored (f_x_tilde). THIS VARIABLE SHOULD BE
* 		ALLOCATED AND INITIALIZED BEFORE CALLING THIS FUNCTION WITH A
* 		SIZE OF "n" TIMES "p=1" 'DOUBLE' MEMORY SPACES.
*
* @param double *A_u - This argument will contain the pointer to a memory
* 		allocated output matrix in which the requested activation
* 		function was applied and stored. "A_u" SHOULD BE ALLOCATED
* 		BEFORE CALLING THIS FUNCTION WITH A SIZE OF "n" TIMES "p=1"
* 		'DOUBLE' MEMORY SPACES.
*
* @param int idx - This argument will contain the value of the GPU thread global
* 		coordinate.
*
* NOTE: RESULT IS STORED IN THE MEMORY ALLOCATED POINTER VARIABLE "A_u".
* 
* @return void
* 
* @author Miranda Meza Cesar
* CREATION DATE: JANUARY 21, 2022
* LAST UPDATE: N/A
*/
__device__ static void getActivationFunction(int activationFunctionToBeUsed, double *u, double *A_u, int idx) {
	// Determine and apply the activation function that was chosen by the implementer.
	// TODO: Use shared memory within the processes of this switch-case but only in the cases in which there is at least a mathematical operation to do. If you use shared memory only to transfer data like in "case 3", you will not obtain better performance results.
	double squareThisValue; // Variable used to store the value that wants to be squared, for performance purposes.
	switch (activationFunctionToBeUsed) {
		case 0: // Rectified Linear Units (ReLU).
			if (u[idx] > 0) {
				A_u[idx] = u[idx];
			} else {
				A_u[idx] = 0;
			}
			break;
		
		case 1: // Hyperbolic tangent (tanh).
			A_u[idx] = (exp(u[idx]) - exp(-u[idx])) / (exp(u[idx]) + exp(-u[idx]));
			break;
		
		case 2: // Logistic function.
			A_u[idx] = 1 / (1 + exp(-u[idx]));
			break;
		
		case 3: // Raise to the 1st power.
			A_u[idx] = u[idx];
			break;
		
		case 4: // Raise to the 2nd power.
			A_u[idx] = u[idx] * u[idx];
			break;
		
		case 5: // Raise to the 3rd power.
			A_u[idx] = u[idx] * u[idx] * u[idx];
			break;
		
		case 6: // Raise to the 4th power.
			squareThisValue = u[idx] * u[idx];
			A_u[idx] = squareThisValue * squareThisValue;
			break;
		
		case 7: // Raise to the 5th power.
			squareThisValue = u[idx] * u[idx];
			A_u[idx] = squareThisValue * squareThisValue * u[idx];
			break;
		
		case 8: // Raise to the 6th power.
			squareThisValue = u[idx] * u[idx];
			A_u[idx] = squareThisValue * squareThisValue * squareThisValue;
			break;
		
		case 9: // 1st order degree exponential.
			A_u[idx] = exp(u[idx]);
			break;
		
		default: // 2nd order degree exponential.
			A_u[idx] = exp(u[idx] * u[idx]);
	}
	return;
}


/**
* The "getDerivateOfActivationFunction()" device static function is
* used to apply a single GPU to calculate and store the derivate of the
* requested activation function that should have been applied to the output of
* the body of a neuron.
*
*
* @param int activationFunctionToBeUsed - This argument will represent the
* 					identifier of the desired activation
* 					function to be used by the neuron during
* 					its training process. Its possible valid
* 					values are the following:
*					0 = Rectified Linear Units (ReLU).
*					1 = Hyperbolic tangent (tanh).
*					2 = Logistic function.
*					3 = Raise to the 1st power.
*					4 = Raise to the 2nd power.
*					5 = Raise to the 3rd power.
*					6 = Raise to the 4th power.
*					7 = Raise to the 5th power.
*					8 = Raise to the 6th power.
*					9 = 1st order degree exponential.
*					10 = 2nd order degree exponential.
*
* @param double *u - This argument will contain the pointer to a memory
*		allocated input matrix, in which the output of the body of a
* 		neuron should be stored (f_x_tilde). THIS VARIABLE SHOULD BE
* 		ALLOCATED AND INITIALIZED BEFORE CALLING THIS FUNCTION WITH A
* 		SIZE OF "n" TIMES "p=1" 'DOUBLE' MEMORY SPACES.
*
* @param double *A_u - This argument will contain the pointer to a memory
* 		allocated output matrix in which the requested activation
* 		function was applied and stored. "A_u" SHOULD BE ALLOCATED AND
* 		INITIALIZED BEFORE CALLING THIS FUNCTION WITH A SIZE OF "n"
* 		TIMES "p=1" 'DOUBLE' MEMORY SPACES.
*
* @param double *dA_u - This argument will contain the pointer to a memory
* 		allocated output matrix in which the derivate of the activation
* 		function was applied and stored. "dA_u" SHOULD BE ALLOCATED
* 		BEFORE CALLING THIS FUNCTION WITH A SIZE OF "n" TIMES "p=1"
* 		'DOUBLE' MEMORY SPACES.
*
* @param int idx - This argument will contain the value of the GPU thread global
* 		coordinate.
*
* NOTE: RESULT IS STORED IN THE MEMORY ALLOCATED POINTER VARIABLE "dA_u".
* 
* @return void
*
* @author Miranda Meza Cesar
* CREATION DATE: JANUARY 21, 2022
* LAST UPDATE: N/A
*/
__device__ static void getDerivateOfActivationFunction(int activationFunctionToBeUsed, double *u, double *A_u, double *dA_u, int idx) {
	// Determine and apply the derivate of the activation function that was chosen by the implementer.
	// TODO: Use shared memory within the processes of this switch-case but only in the cases in which there is at least a mathematical operation to do. If you use shared memory only to transfer data like in "case 3", you will not obtain better performance results.
	double squareThisValue; // Variable used to store the value that wants to be squared, for performance purposes.
	switch (activationFunctionToBeUsed) {
		case 0: // Rectified Linear Units (ReLU).
			if (u[idx] > 0) {
				dA_u[idx] = 1;
			} else {
				dA_u[idx] = 0;
			}
			break;
		
		case 1: // Hyperbolic tangent (tanh).
			dA_u[idx] = 1 - A_u[idx] * A_u[idx];
			break;
		
		case 2: // Logistic function.
			dA_u[idx] = A_u[idx] * (1 - A_u[idx]);
			break;
		
		case 3: // Raise to the 1st power.
			dA_u[idx] = 1;
			break;
		
		case 4: // Raise to the 2nd power.
			dA_u[idx] = 2*u[idx];
			break;
		
		case 5: // Raise to the 3rd power.
			dA_u[idx] = 3 * u[idx] * u[idx];
			break;
		
		case 6: // Raise to the 4th power.
			dA_u[idx] = 4 * u[idx] * u[idx] * u[idx];
			break;
		
		case 7: // Raise to the 5th power.
			squareThisValue = u[idx] * u[idx];
			dA_u[idx] = 5 * squareThisValue * squareThisValue;
			break;
		
		case 8: // Raise to the 6th power.
			squareThisValue = u[idx] * u[idx];
			dA_u[idx] = 6 * squareThisValue * squareThisValue * u[idx];
			break;
		
		case 9: // 1st order degree exponential.
			dA_u[idx] = A_u[idx];
			break;
		
		default: // 2nd order degree exponential.
			dA_u[idx] = 2 * u[idx] * A_u[idx];
	}
	return;
}


/**
* The "getNeuronAdjustedCoefficientOfDetermination_singleGPUPart1()" device
* static function is used to apply a single GPU to make several calculations
* required to obtain the adjusted R-squared evaluation metric with respect to
* the actual data of the system under study "Y" and the currently predicted
* output made by the neuron "A_u" (with the intention of applying the Parallel
* Reduction strategy to it on another process, external to this function).
* 
*
* @param double *Y - This argument will contain the pointer to a memory
* 		allocated output matrix, representing the real data of the
*		system under study. THIS VARIABLE SHOULD BE ALLOCATED AND
* 		INITIALIZED BEFORE CALLING THIS FUNCTION WITH A SIZE OF "n"
* 		TIMES "p=1" 'DOUBLE' MEMORY SPACES.
*
* @param double *A_u - This argument will contain the pointer to a memory
* 		allocated output matrix in which the requested activation
* 		function was applied and stored. "A_u" SHOULD BE ALLOCATED AND
* 		INITIALIZED BEFORE CALLING THIS FUNCTION WITH A SIZE OF "n"
* 		TIMES "p=1" 'DOUBLE' MEMORY SPACES.
*
* @param double *accuracyTerm1 - This argument will contain the pointer to a
* 			memory allocated matrix that will contain all the
* 			calculations required to obtained the SSE value (with the
* 			intention of applying the Parallel Reduction strategy to
* 			it on another process, external to this function). IT IS
* 			INDISPENSABLE THAT THIS VARIABLE IS ALLOCATED BEFORE
* 			CALLING THIS FUNCTION WITH A VARIABLE SIZE OF "1" TIMES
* 			"n" 'DOUBLE' MEMORY SPACES.
*
* @param double *accuracyTerm2 - This argument will contain the pointer to a
* 			memory allocated matrix that will contain all the
* 			calculations required to obtained the sum of all the
* 			values contained in the argument pointer variable "Y"
* 			(with the intention of applying the Parallel Reduction
* 			strategy to it on another process, external to this
* 			function). IT IS INDISPENSABLE THAT THIS VARIABLE IS
* 			ALLOCATED BEFORE CALLING THIS FUNCTION WITH A VARIABLE
* 			SIZE OF "1" TIMES "n" 'DOUBLE' MEMORY SPACES.
*
* @param int idx - This argument will contain the value of the GPU thread global
* 		coordinate.
*
*
* NOTE: RESULTS ARE STORED IN "accuracyTerm1" and "accuracyTerm2".
* 
* @return void
*
* @author Miranda Meza Cesar
* CREATION DATE: JANUARY 24, 2022
* LAST UPDATE: JANUARY 25, 2022
*/
__device__ static void getNeuronAdjustedCoefficientOfDetermination_singleGPUPart1(double *Y, double *A_u, double *accuracyTerm1, double *accuracyTerm2, int idx) {
	// We obtain the GPU thread local coordinate.
	int tid = threadIdx.x;
	
	// We declare and initialize the shared memory of the GPU that will be used.
	extern __shared__ double sharedMem[]; // We declare the shared memory that we will use for each block.
	// NOTE: Each GPU thread is storing data such that their local address (tid) will represent the identifier of the row number in which they will write data in the shared memory. Moreover, each thread will have assigned 2 columns per row.
	int column1 = tid * 2;
	int column2 = 1 + column1;
	sharedMem[column1] = Y[idx];
	sharedMem[column2] = A_u[idx];
	
	// We obtain and store all the GPU threads contibutions to calculate the sum of the real output matrix.
	accuracyTerm2[idx] = sharedMem[column1]; // We temporarily store the sum of the real output matrix in the argument pointer variable "accuracyTerm2", for performance purposes.
	
	// We obtain and store all the GPU threads contibutions to calculate the SSE value.
	sharedMem[column1] = sharedMem[column1] - sharedMem[column2]; // real output matrix - predicted output matrix
	sharedMem[column2] = sharedMem[column1] * sharedMem[column1]; // We square the value that was previously obtained.
	accuracyTerm1[idx] = sharedMem[column2]; // We temporarly store the SSE values in the argument pointer variable "accuracyTerm1", for performance purposes.
	
	return;
}


/**
* The "getParallelReduction()" global static function is used to employ a single
* GPU to apply the Parallel Reduction strategy to the argument variable
* "termToBeReduced".
* 
*
* @param double *termToBeReduced - This argument will contain the pointer to a
* 				memory allocated variable on which it is desired
* 				to apply the Parallel Reduction strategy.
*
* @param double *reducedAccuracyTerm - This argument will contain the pointer to
* 				a memory allocated variable in which we will
* 				store the result of having applied the Parallel
* 				Reduction strategy to the argument pointer
* 				variable "termToBeReduced".
*
* @param int n - This argument will represent the total number of samples (rows)
* 		that the input matrix has, with which the output data was
*		obtained.
*
* @param int numberOfUnrollingLoop - This argument is used to represent the
* 				number of unrolling loops that the algorithm will
* 				use when applying the Parallel Reduction
*				Strategy.
*
* NOTE: RESULTS ARE STORED IN THE MEMORY ALLOCATED POINTER VARIABLES
*       "reducedAccuracyTerm".
* 
* @return void
*
* @author Miranda Meza Cesar
* CREATION DATE: JANUARY 24, 2022
* LAST UPDATE: N/A
*/
__global__ static void getParallelReduction(double *termToBeReduced, double *reducedAccuracyTerm, int n, int numberOfUnrollingLoop) {
	// We declare the variables that will be given a value through the next case code.
	int idx; // Variable used to store the GPU thread global coordinate.
	int tid = threadIdx.x; // Variable used to store the GPU thread local coordinate.
	double *idata; // Variable used to convert the pointer of interest from "termToBeReduced" to be the origin pointer of "idata".
	double unroll1; // Variable used in the "Unrolling Loop Strategy" that applies, if any.
	double unroll2; // Variable used in the "Unrolling Loop Strategy" that applies, if any.
	double unroll3; // Variable used in the "Unrolling Loop Strategy" that applies, if any.
	double unroll4; // Variable used in the "Unrolling Loop Strategy" that applies, if any.
	double unroll5; // Variable used in the "Unrolling Loop Strategy" that applies, if any.
	double unroll6; // Variable used in the "Unrolling Loop Strategy" that applies, if any.
	double unroll7; // Variable used in the "Unrolling Loop Strategy" that applies, if any.
	double unroll8; // Variable used in the "Unrolling Loop Strategy" that applies, if any.
	double unroll9; // Variable used in the "Unrolling Loop Strategy" that applies, if any.
	double unroll10; // Variable used in the "Unrolling Loop Strategy" that applies, if any.
	
	// Parallel Reduction Strategy: Unrolling Strategy process.
	switch (numberOfUnrollingLoop) {
		case 10: // "Unrolling10 Strategy": Unrolling 10 times, but only with the GPU threads that are within the boundary.
			idx = threadIdx.x + 10*blockIdx.x*blockDim.x;
			idata = termToBeReduced;
			if ((idx + 9*blockDim.x) < n) {
				unroll1 = idata[idx]; idata += blockDim.x;
				unroll2 = idata[idx]; idata += blockDim.x;
				unroll3 = idata[idx]; idata += blockDim.x;
				unroll4 = idata[idx]; idata += blockDim.x;
				unroll5 = idata[idx]; idata += blockDim.x;
				unroll6 = idata[idx]; idata += blockDim.x;
				unroll7 = idata[idx]; idata += blockDim.x;
				unroll8 = idata[idx]; idata += blockDim.x;
				unroll9 = idata[idx]; idata += blockDim.x;
				unroll10 = idata[idx];
				termToBeReduced[idx] = unroll1 + unroll2 + unroll3 + unroll4 + unroll5 + unroll6 + unroll7 + unroll8 + unroll9 + unroll10;
			}
			idata = termToBeReduced + 10*blockIdx.x*blockDim.x;
		break;
		
		case 9: // "Unrolling9 Strategy": Unrolling 9 times, but only with the GPU threads that are within the boundary.
			idx = threadIdx.x + 9*blockIdx.x*blockDim.x;
			idata = termToBeReduced;
			if ((idx + 8*blockDim.x) < n) {
				unroll1 = idata[idx]; idata += blockDim.x;
				unroll2 = idata[idx]; idata += blockDim.x;
				unroll3 = idata[idx]; idata += blockDim.x;
				unroll4 = idata[idx]; idata += blockDim.x;
				unroll5 = idata[idx]; idata += blockDim.x;
				unroll6 = idata[idx]; idata += blockDim.x;
				unroll7 = idata[idx]; idata += blockDim.x;
				unroll8 = idata[idx]; idata += blockDim.x;
				unroll9 = idata[idx];
				termToBeReduced[idx] = unroll1 + unroll2 + unroll3 + unroll4 + unroll5 + unroll6 + unroll7 + unroll8 + unroll9;
			}
			idata = termToBeReduced + 9*blockIdx.x*blockDim.x;
		break;
		
		case 8: // "Unrolling8 Strategy": Unrolling 8 times, but only with the GPU threads that are within the boundary.
			idx = threadIdx.x + 8*blockIdx.x*blockDim.x;
			idata = termToBeReduced;
			if ((idx + 7*blockDim.x) < n) {
				unroll1 = idata[idx]; idata += blockDim.x;
				unroll2 = idata[idx]; idata += blockDim.x;
				unroll3 = idata[idx]; idata += blockDim.x;
				unroll4 = idata[idx]; idata += blockDim.x;
				unroll5 = idata[idx]; idata += blockDim.x;
				unroll6 = idata[idx]; idata += blockDim.x;
				unroll7 = idata[idx]; idata += blockDim.x;
				unroll8 = idata[idx];
				termToBeReduced[idx] = unroll1 + unroll2 + unroll3 + unroll4 + unroll5 + unroll6 + unroll7 + unroll8;
			}
			idata = termToBeReduced + 8*blockIdx.x*blockDim.x;
		break;
		
		case 7: // "Unrolling7 Strategy": Unrolling 7 times, but only with the GPU threads that are within the boundary.
			idx = threadIdx.x + 7*blockIdx.x*blockDim.x;
			idata = termToBeReduced;
			if ((idx + 6*blockDim.x) < n) {
				unroll1 = idata[idx]; idata += blockDim.x;
				unroll2 = idata[idx]; idata += blockDim.x;
				unroll3 = idata[idx]; idata += blockDim.x;
				unroll4 = idata[idx]; idata += blockDim.x;
				unroll5 = idata[idx]; idata += blockDim.x;
				unroll6 = idata[idx]; idata += blockDim.x;
				unroll7 = idata[idx];
				termToBeReduced[idx] = unroll1 + unroll2 + unroll3 + unroll4 + unroll5 + unroll6 + unroll7;
			}
			idata = termToBeReduced + 7*blockIdx.x*blockDim.x;
		break;
		
		case 6: // "Unrolling6 Strategy": Unrolling 6 times, but only with the GPU threads that are within the boundary.
			idx = threadIdx.x + 6*blockIdx.x*blockDim.x;
			idata = termToBeReduced;
			if ((idx + 5*blockDim.x) < n) {
				unroll1 = idata[idx]; idata += blockDim.x;
				unroll2 = idata[idx]; idata += blockDim.x;
				unroll3 = idata[idx]; idata += blockDim.x;
				unroll4 = idata[idx]; idata += blockDim.x;
				unroll5 = idata[idx]; idata += blockDim.x;
				unroll6 = idata[idx];
				termToBeReduced[idx] = unroll1 + unroll2 + unroll3 + unroll4 + unroll5 + unroll6;
			}
			idata = termToBeReduced + 6*blockIdx.x*blockDim.x;
		break;
		
		case 5: // "Unrolling5 Strategy": Unrolling 5 times, but only with the GPU threads that are within the boundary.
			idx = threadIdx.x + 5*blockIdx.x*blockDim.x;
			idata = termToBeReduced;
			if ((idx + 4*blockDim.x) < n) {
				unroll1 = idata[idx]; idata += blockDim.x;
				unroll2 = idata[idx]; idata += blockDim.x;
				unroll3 = idata[idx]; idata += blockDim.x;
				unroll4 = idata[idx]; idata += blockDim.x;
				unroll5 = idata[idx];
				termToBeReduced[idx] = unroll1 + unroll2 + unroll3 + unroll4 + unroll5;
			}
			idata = termToBeReduced + 5*blockIdx.x*blockDim.x;
		break;
		
		case 4: // "Unrolling4 Strategy": Unrolling 4 times, but only with the GPU threads that are within the boundary.
			idx = threadIdx.x + 4*blockIdx.x*blockDim.x;
			idata = termToBeReduced;
			if ((idx + 3*blockDim.x) < n) {
				unroll1 = idata[idx]; idata += blockDim.x;
				unroll2 = idata[idx]; idata += blockDim.x;
				unroll3 = idata[idx]; idata += blockDim.x;
				unroll4 = idata[idx];
				termToBeReduced[idx] = unroll1 + unroll2 + unroll3 + unroll4;
			}
			idata = termToBeReduced + 4*blockIdx.x*blockDim.x;
		break;
		
		case 3: // "Unrolling3 Strategy": Unrolling 3 times, but only with the GPU threads that are within the boundary.
			idx = threadIdx.x + 3*blockIdx.x*blockDim.x;
			idata = termToBeReduced;
			if ((idx + 2*blockDim.x) < n) {
				unroll1 = idata[idx]; idata += blockDim.x;
				unroll2 = idata[idx]; idata += blockDim.x;
				unroll3 = idata[idx];
				termToBeReduced[idx] = unroll1 + unroll2 + unroll3;
			}
			idata = termToBeReduced + 3*blockIdx.x*blockDim.x;
		break;
		
		case 2: // "Unrolling2 Strategy": Unrolling 2 times, but only with the GPU threads that are within the boundary.
			idx = threadIdx.x + 2*blockIdx.x*blockDim.x;
			idata = termToBeReduced + 2*blockIdx.x*blockDim.x;
			if ((idx + blockDim.x) < n) {
				unroll1 = termToBeReduced[idx];
				unroll2 = termToBeReduced[idx + blockDim.x];
				termToBeReduced[idx] = unroll1 + unroll2;
			}
		break;
		
		default: // No "Unrolling Strategy" will be applied.
			idx = threadIdx.x + blockIdx.x*blockDim.x;
			idata = termToBeReduced + blockIdx.x*blockDim.x;
	}
	__syncthreads(); // We synchronize all threads within the same block.
	
	// Parallel Reduction Strategy: Unrolling Warp process with shared memory.
	extern __shared__ double sharedMem[]; // We declare the shared memory that we will use for each block.
	sharedMem[tid] = idata[tid];
	__syncthreads(); // We synchronize all threads within the same block.
	if (tid < 16) {
		sharedMem[tid] += sharedMem[tid+16];__syncthreads(); // We synchronize all threads within the same block.
		sharedMem[tid] += sharedMem[tid+8];__syncthreads(); // We synchronize all threads within the same block.
		sharedMem[tid] += sharedMem[tid+4];__syncthreads(); // We synchronize all threads within the same block.
		sharedMem[tid] += sharedMem[tid+2];__syncthreads(); // We synchronize all threads within the same block.
		sharedMem[tid] += sharedMem[tid+1];__syncthreads(); // We synchronize all threads within the same block.
	}
	
	// Store the final result corresponding to the current block.
	if (tid == 0) reducedAccuracyTerm[blockIdx.x] = sharedMem[0];
	
	return;
}


/**
* The "getDeviceParallelReduction()" device static function is used to employ a
* single GPU to apply the Parallel Reduction strategy to the argument variable
* "termToBeReduced".
*
*
* @param double *termToBeReduced - This argument will contain the pointer to a
* 				memory allocated variable on which it is desired
* 				to apply the Parallel Reduction strategy.
*
* @param double *reducedAccuracyTerm - This argument will contain the pointer to
* 				a memory allocated variable in which we will
* 				store the result of having applied the Parallel
* 				Reduction strategy to the argument pointer
* 				variable "termToBeReduced".
*
* @param int n - This argument will represent the total number of samples (rows)
* 		that the input matrix has, with which the output data was
*		obtained.
*
* @param int numberOfUnrollingLoop - This argument is used to represent the
* 				number of unrolling loops that the algorithm will
* 				use when applying the Parallel Reduction
*				Strategy.
*
* NOTE: RESULTS ARE STORED IN THE MEMORY ALLOCATED POINTER VARIABLES
*       "reducedAccuracyTerm".
* 
* @return void
*
* @author Miranda Meza Cesar
* CREATION DATE: JANUARY 24, 2022
* LAST UPDATE: N/A
*/
__device__ static void getDeviceParallelReduction(double *termToBeReduced, double *reducedAccuracyTerm, int n, int numberOfUnrollingLoop) {
	// We declare the variables that will be given a value through the next case code.
	int idx; // Variable used to store the GPU thread global coordinate.
	int tid = threadIdx.x; // Variable used to store the GPU thread local coordinate.
	double *idata; // Variable used to convert the pointer of interest from "termToBeReduced" to be the origin pointer of "idata".
	double unroll1; // Variable used in the "Unrolling Loop Strategy" that applies, if any.
	double unroll2; // Variable used in the "Unrolling Loop Strategy" that applies, if any.
	double unroll3; // Variable used in the "Unrolling Loop Strategy" that applies, if any.
	double unroll4; // Variable used in the "Unrolling Loop Strategy" that applies, if any.
	double unroll5; // Variable used in the "Unrolling Loop Strategy" that applies, if any.
	double unroll6; // Variable used in the "Unrolling Loop Strategy" that applies, if any.
	double unroll7; // Variable used in the "Unrolling Loop Strategy" that applies, if any.
	double unroll8; // Variable used in the "Unrolling Loop Strategy" that applies, if any.
	double unroll9; // Variable used in the "Unrolling Loop Strategy" that applies, if any.
	double unroll10; // Variable used in the "Unrolling Loop Strategy" that applies, if any.
	
	// Parallel Reduction Strategy: Unrolling Strategy process.
	switch (numberOfUnrollingLoop) {
		case 10: // "Unrolling10 Strategy": Unrolling 10 times, but only with the GPU threads that are within the boundary.
			idx = threadIdx.x + 10*blockIdx.x*blockDim.x;
			idata = termToBeReduced;
			if ((idx + 9*blockDim.x) < n) {
				unroll1 = idata[idx]; idata += blockDim.x;
				unroll2 = idata[idx]; idata += blockDim.x;
				unroll3 = idata[idx]; idata += blockDim.x;
				unroll4 = idata[idx]; idata += blockDim.x;
				unroll5 = idata[idx]; idata += blockDim.x;
				unroll6 = idata[idx]; idata += blockDim.x;
				unroll7 = idata[idx]; idata += blockDim.x;
				unroll8 = idata[idx]; idata += blockDim.x;
				unroll9 = idata[idx]; idata += blockDim.x;
				unroll10 = idata[idx];
				termToBeReduced[idx] = unroll1 + unroll2 + unroll3 + unroll4 + unroll5 + unroll6 + unroll7 + unroll8 + unroll9 + unroll10;
			}
			idata = termToBeReduced + 10*blockIdx.x*blockDim.x;
		break;
		
		case 9: // "Unrolling9 Strategy": Unrolling 9 times, but only with the GPU threads that are within the boundary.
			idx = threadIdx.x + 9*blockIdx.x*blockDim.x;
			idata = termToBeReduced;
			if ((idx + 8*blockDim.x) < n) {
				unroll1 = idata[idx]; idata += blockDim.x;
				unroll2 = idata[idx]; idata += blockDim.x;
				unroll3 = idata[idx]; idata += blockDim.x;
				unroll4 = idata[idx]; idata += blockDim.x;
				unroll5 = idata[idx]; idata += blockDim.x;
				unroll6 = idata[idx]; idata += blockDim.x;
				unroll7 = idata[idx]; idata += blockDim.x;
				unroll8 = idata[idx]; idata += blockDim.x;
				unroll9 = idata[idx];
				termToBeReduced[idx] = unroll1 + unroll2 + unroll3 + unroll4 + unroll5 + unroll6 + unroll7 + unroll8 + unroll9;
			}
			idata = termToBeReduced + 9*blockIdx.x*blockDim.x;
		break;
		
		case 8: // "Unrolling8 Strategy": Unrolling 8 times, but only with the GPU threads that are within the boundary.
			idx = threadIdx.x + 8*blockIdx.x*blockDim.x;
			idata = termToBeReduced;
			if ((idx + 7*blockDim.x) < n) {
				unroll1 = idata[idx]; idata += blockDim.x;
				unroll2 = idata[idx]; idata += blockDim.x;
				unroll3 = idata[idx]; idata += blockDim.x;
				unroll4 = idata[idx]; idata += blockDim.x;
				unroll5 = idata[idx]; idata += blockDim.x;
				unroll6 = idata[idx]; idata += blockDim.x;
				unroll7 = idata[idx]; idata += blockDim.x;
				unroll8 = idata[idx];
				termToBeReduced[idx] = unroll1 + unroll2 + unroll3 + unroll4 + unroll5 + unroll6 + unroll7 + unroll8;
			}
			idata = termToBeReduced + 8*blockIdx.x*blockDim.x;
		break;
		
		case 7: // "Unrolling7 Strategy": Unrolling 7 times, but only with the GPU threads that are within the boundary.
			idx = threadIdx.x + 7*blockIdx.x*blockDim.x;
			idata = termToBeReduced;
			if ((idx + 6*blockDim.x) < n) {
				unroll1 = idata[idx]; idata += blockDim.x;
				unroll2 = idata[idx]; idata += blockDim.x;
				unroll3 = idata[idx]; idata += blockDim.x;
				unroll4 = idata[idx]; idata += blockDim.x;
				unroll5 = idata[idx]; idata += blockDim.x;
				unroll6 = idata[idx]; idata += blockDim.x;
				unroll7 = idata[idx];
				termToBeReduced[idx] = unroll1 + unroll2 + unroll3 + unroll4 + unroll5 + unroll6 + unroll7;
			}
			idata = termToBeReduced + 7*blockIdx.x*blockDim.x;
		break;
		
		case 6: // "Unrolling6 Strategy": Unrolling 6 times, but only with the GPU threads that are within the boundary.
			idx = threadIdx.x + 6*blockIdx.x*blockDim.x;
			idata = termToBeReduced;
			if ((idx + 5*blockDim.x) < n) {
				unroll1 = idata[idx]; idata += blockDim.x;
				unroll2 = idata[idx]; idata += blockDim.x;
				unroll3 = idata[idx]; idata += blockDim.x;
				unroll4 = idata[idx]; idata += blockDim.x;
				unroll5 = idata[idx]; idata += blockDim.x;
				unroll6 = idata[idx];
				termToBeReduced[idx] = unroll1 + unroll2 + unroll3 + unroll4 + unroll5 + unroll6;
			}
			idata = termToBeReduced + 6*blockIdx.x*blockDim.x;
		break;
		
		case 5: // "Unrolling5 Strategy": Unrolling 5 times, but only with the GPU threads that are within the boundary.
			idx = threadIdx.x + 5*blockIdx.x*blockDim.x;
			idata = termToBeReduced;
			if ((idx + 4*blockDim.x) < n) {
				unroll1 = idata[idx]; idata += blockDim.x;
				unroll2 = idata[idx]; idata += blockDim.x;
				unroll3 = idata[idx]; idata += blockDim.x;
				unroll4 = idata[idx]; idata += blockDim.x;
				unroll5 = idata[idx];
				termToBeReduced[idx] = unroll1 + unroll2 + unroll3 + unroll4 + unroll5;
			}
			idata = termToBeReduced + 5*blockIdx.x*blockDim.x;
		break;
		
		case 4: // "Unrolling4 Strategy": Unrolling 4 times, but only with the GPU threads that are within the boundary.
			idx = threadIdx.x + 4*blockIdx.x*blockDim.x;
			idata = termToBeReduced;
			if ((idx + 3*blockDim.x) < n) {
				unroll1 = idata[idx]; idata += blockDim.x;
				unroll2 = idata[idx]; idata += blockDim.x;
				unroll3 = idata[idx]; idata += blockDim.x;
				unroll4 = idata[idx];
				termToBeReduced[idx] = unroll1 + unroll2 + unroll3 + unroll4;
			}
			idata = termToBeReduced + 4*blockIdx.x*blockDim.x;
		break;
		
		case 3: // "Unrolling3 Strategy": Unrolling 3 times, but only with the GPU threads that are within the boundary.
			idx = threadIdx.x + 3*blockIdx.x*blockDim.x;
			idata = termToBeReduced;
			if ((idx + 2*blockDim.x) < n) {
				unroll1 = idata[idx]; idata += blockDim.x;
				unroll2 = idata[idx]; idata += blockDim.x;
				unroll3 = idata[idx];
				termToBeReduced[idx] = unroll1 + unroll2 + unroll3;
			}
			idata = termToBeReduced + 3*blockIdx.x*blockDim.x;
		break;
		
		case 2: // "Unrolling2 Strategy": Unrolling 2 times, but only with the GPU threads that are within the boundary.
			idx = threadIdx.x + 2*blockIdx.x*blockDim.x;
			idata = termToBeReduced + 2*blockIdx.x*blockDim.x;
			if ((idx + blockDim.x) < n) {
				unroll1 = termToBeReduced[idx];
				unroll2 = termToBeReduced[idx + blockDim.x];
				termToBeReduced[idx] = unroll1 + unroll2;
			}
		break;
		
		default: // No "Unrolling Strategy" will be applied.
			idx = threadIdx.x + blockIdx.x*blockDim.x;
			idata = termToBeReduced + blockIdx.x*blockDim.x;
	}
	__syncthreads(); // We synchronize all threads within the same block.
	
	// Parallel Reduction Strategy: Unrolling Warp process with shared memory.
	extern __shared__ double sharedMem[]; // We declare the shared memory that we will use for each block.
	sharedMem[tid] = idata[tid];
	__syncthreads(); // We synchronize all threads within the same block.
	if (tid < 16) {
		sharedMem[tid] += sharedMem[tid+16];__syncthreads(); // We synchronize all threads within the same block.
		sharedMem[tid] += sharedMem[tid+8];__syncthreads(); // We synchronize all threads within the same block.
		sharedMem[tid] += sharedMem[tid+4];__syncthreads(); // We synchronize all threads within the same block.
		sharedMem[tid] += sharedMem[tid+2];__syncthreads(); // We synchronize all threads within the same block.
		sharedMem[tid] += sharedMem[tid+1];__syncthreads(); // We synchronize all threads within the same block.
	}
	
	// Store the final result corresponding to the current block.
	if (tid == 0) reducedAccuracyTerm[blockIdx.x] = sharedMem[0];
	
	return;
}


/**
* The "getNeuronAdjustedCoefficientOfDetermination_singleGPUvoidPart2()" global static
* function is used to apply a single GPU to calculate "f(\tilde{x})", A(u),
* dA(u) and the first part of the accuracy terms calculations.
* 
* @param double *Y - This argument will contain the pointer to a memory
* 		allocated output matrix, representing the real data of the
*		system under study. THIS VARIABLE SHOULD BE ALLOCATED AND
* 		INITIALIZED BEFORE CALLING THIS FUNCTION WITH A SIZE OF "n"
* 		TIMES "p=1" 'DOUBLE' MEMORY SPACES.
* 
* @param int n - This argument will represent the total number of samples (rows)
* 		that the input matrix has, with which the output data was
*		obtained.
*
* @param double *accuracyTerm1 - This argument will contain the pointer to a
* 			memory allocated matrix that will contain all the
* 			calculations required to obtained the SSE value (with the
* 			intention of applying the Parallel Reduction strategy to
* 			it on another process, external to this function). IT IS
* 			INDISPENSABLE THAT THIS VARIABLE IS ALLOCATED BEFORE
* 			CALLING THIS FUNCTION WITH A VARIABLE SIZE OF "1" TIMES
* 			"n" 'DOUBLE' MEMORY SPACES.
*
* @param double *reducedAccuracyTerm2 - This argument will contain the pointer
* 				to a memory allocated matrix that will contain
* 				the mean of the real output matrix ("Y") in the
* 				address with the identifier "0" in it. IT IS
* 				INDISPENSABLE THAT THIS VARIABLE IS ALLOCATED AND
* 				INITIALIZED BEFORE CALLING THIS FUNCTION WITH A
* 				VARIABLE SIZE OF "1" TIMES "grid size of the GPU
* 				Kernel" 'DOUBLE' MEMORY SPACES.
* 
*
* NOTE: RESULTS ARE STORED IN "accuracyTerm1".
* 
* @return void
*
* @author Miranda Meza Cesar
* CREATION DATE: JANUARY 24, 2022
* LAST UPDATE: JANUARY 25, 2022
*/
__global__ static void getNeuronAdjustedCoefficientOfDetermination_singleGPUvoidPart2(double *Y, int n, double *accuracyTerm1, double *reducedAccuracyTerm2) {
	// We obtain the GPU threads coordinates.
	int tid = threadIdx.x; // We obtain the GPU thread local coordinate
	int idx = threadIdx.x + blockIdx.x * blockDim.x; // We obtain the GPU thread global coordinate.
	
	// If the current GPU thread is within boundary, then proceed to work with the task. Otherwise, conclude your operation.
	if (idx < n) {
		// We declare and initialize the shared memory of the GPU that will be used.
		extern __shared__ double sharedMem[]; // We declare the shared memory that we will use for each block.
		// NOTE: Each GPU thread is storing data such that their local address (tid) will represent the identifier of the row number in which they will write data in the shared memory. Moreover, each thread will have assigned 2 columns per row.
		int column1 = tid * 2;
		int column2 = 1 + column1;
		sharedMem[column1] = Y[idx];
		sharedMem[column2] = reducedAccuracyTerm2[0];
		
		// We get the MSSE value.
		sharedMem[column1] = sharedMem[column1] - sharedMem[column2];
		sharedMem[column2] = sharedMem[column1] * sharedMem[column1];
		accuracyTerm1[idx] = sharedMem[column2];
	}
	
	return;
}


/**
* The "predictSingleNeuronDNN_singleGPU()" function is used to apply a single
* GPU to make the predictions of the requested input values (X) by applying the
* simple linear equation system with the specified coefficient values (b). The
* predicted values will be stored in the argument pointer variable "Y_hat".
* 
* @param struct singleNeuronDnnStruct_singleGPU *neuron - This argument will
* 					contain the pointer to a struct variable
* 					that should contain all the information
* 					required in order to be able to create
* 					and make an artificial neuron. Its
* 					accessible inner elements are described
* 					in the list showed in the commented
* 					documentation of the function
* 					"getSingleNeuronDNN_singleGPU()".
*
* @param double *Y_hat - This argument will contain the pointer to a memory
* 			allocated output matrix, representing the predicted data
* 			of the system under study. THIS VARIABLE SHOULD BE
* 			ALLOCATED BEFORE CALLING THIS FUNCTION WITH A SIZE OF "n"
* 			TIMES "p=1" 'DOUBLE' MEMORY SPACES. The results will be
* 			stored in the same order as the input data given such
* 			that the first sample will be stored in the row with
* 			index "0" and the last sample in the row with index "n".
*
* NOTE: RESULT IS STORED IN THE MEMORY ALLOCATED POINTER VARIABLE "Y_hat".
* 
* @return void
*
* @author Miranda Meza Cesar
* CREATION DATE: JANUARY 24, 2022
* LAST UPDATE: JANUARY 25, 2022
*/
void predictSingleNeuronDNN_singleGPU(struct singleNeuronDnnStruct_singleGPU *neuron, double *Y_hat) {
	// If the requested GPU (device) is less than zero, then emit an error message and terminate the program. Otherwise, continue with the program.
	if (neuron->gpuDevice < 0) {
		printf("\nERROR: The identifier of the requested GPU (device) must be equal or greater than 0.\n");
		exit(1);
	}
	// If the machine learning samples are less than value of one, then emit an error message and terminate the program. Otherwise, continue with the program.
	if (neuron->n < 1) {
		printf("\nERROR: The machine learning samples must be equal or greater than 1 for this particular algorithm.\n");
		exit(1);
	}
	// If the machine learning features are less than the value of one, then emit an error message and terminate the program. Otherwise, continue with the program.
	if (neuron->m < 1) {
		printf("\nERROR: The machine learning features (independent variables) must be equal or greater than 1 for this particular algorithm.\n");
		exit(1);
	}
	// If the output of the system under study is different than the value of one, then emit an error message and terminate the program. Otherwise, continue with the program.
	if (neuron->p != 1) {
		printf("\nERROR: The outputs of the system under study must be equal to 1 for this particular algorithm.\n");
		exit(1);
	}
	// If the identifier assigned to "neuron->activationFunctionToBeUsed" is not in the range of 0 and 11, then emit an error message and terminate the program. Otherwise, continue with the program.
	if ((neuron->activationFunctionToBeUsed>11) && (neuron->activationFunctionToBeUsed<0)) {
		printf("\nERROR: The defined activation function identifier assigned to \"activationFunctionToBeUsed\" in the struct of \"singleNeuronDnnStruct\" must be a whole value in the range of 0 to 11. Please add a valid identifier number to it.\n");
		exit(1);
	}
	// If the flag "neuron->isClassification" has a value different of 0 and 1, then emit an error message and terminate the program. Otherwise, continue with the program.
	if ((neuron->isClassification!=0) && (neuron->isClassification!=1)) {
		printf("\nERROR: The defined value for the flag \"isClassification\" in the struct of \"singleNeuronDnnStruct\" can only have a value of either 0 or 1. Please add a valid value to it.\n");
		exit(1);
	}
	
	
	// ------- SELECTION AND INITIALIZATION OF THE DESIRED GPU ------- //
	// We selected the GPU desired by the implementer and inform in the terminal the name of such GPU.
	hipDeviceProp_t gpuProperties;
	CHECK(hipGetDeviceProperties(&gpuProperties, neuron->gpuDevice)); // We obtain the details of the GPU that was defined by the implementer.
	printf("\nThe GPU (device) %d: %s, has been selected by the CenyML library.\n", neuron->gpuDevice, gpuProperties.name);
	CHECK(hipSetDevice(neuron->gpuDevice)); // We select the GPU that was requested by the implementer.
	
	// Set up the execution configurations that will be assigned to the selected GPU.
	dim3 block_32x_1y(32, 1); // We define the number of GPU threads per block.
	dim3 grid_n((neuron->n + block_32x_1y.x - 1) / block_32x_1y.x, 1); // We define the number of blocks that our GPU will manage.
	
	// We configure the shared memory of the current GPU.
	hipSharedMemConfig pConfig = hipSharedMemBankSizeEightByte; // We create a hipSharedMemConfig type variable to store in it the configuration of 8-byte mode for shared memory in the GPU.
	hipDeviceSetSharedMemConfig(pConfig); // We set the 8-byte mode for shared memory in the selected GPU.
	
	
	// --------------- PREPROCESSING OF THE INPUT DATA --------------- //
	// We create the pointers to the data that the selected GPU will require.
	double *d_X; // This pointer variable is used to store the data from "neuron->X" into the selected GPU.
	double *d_w_new; // This pointer variable is used to store the data from "neuron->w_new" into the selected GPU.
	double *d_f_x_tilde; // This pointer variable is used to store the output of the body of the neuron in the selected GPU.
	double *d_A_u; // This pointer variable is used to store the output of the application of the chosen activation function in the selected GPU.
	
	// We allocate the required memory in the selected GPU.
	int nDoubles_Bytes = neuron->n*sizeof(double); // This variable stores the number of bytes to allocate "n" bytes of double type, for performance purposes.
	CHECK(hipMalloc((void **) &d_X, neuron->m*nDoubles_Bytes));
	int w_new_Bytes = (neuron->m+1)*sizeof(double); // This variable stores the number of bytes to allocate the new weight values that will be obtained each epoch of the training process, for performance purposes.
	CHECK(hipMalloc((void **) &d_w_new, w_new_Bytes));
	CHECK(hipMalloc((void **) &d_f_x_tilde, nDoubles_Bytes));
	CHECK(hipMalloc((void **) &d_A_u, nDoubles_Bytes));
	
	// We transfer the required data from the CPU to the selected GPU.
	CHECK(hipMemcpy(d_w_new, neuron->w_best, w_new_Bytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_X, neuron->X, (neuron->n*neuron->m*sizeof(double)), hipMemcpyHostToDevice));
	
	
	// --------------- DATA PREDICTION PROCESS --------------- //
	// We obtain the requested predictions from the artificial neuron model.
	getPredictSingleNeuronDNN_singleGPU <<< grid_n, block_32x_1y, (3*32*sizeof(double)) >>> (d_X, d_w_new, neuron->n, neuron->m, neuron->activationFunctionToBeUsed, neuron->isClassification, neuron->threshold, neuron->desiredValueForGroup1, neuron->desiredValueForGroup2, d_f_x_tilde, d_A_u);
	
	// We transfer the predicted data from the GPU to the CPU, to the argument variable "Y_hat".
	CHECK(hipMemcpy(Y_hat, d_A_u, nDoubles_Bytes, hipMemcpyDeviceToHost));
	
	
	// Before terminating this function, we free the GPU and CPU allocated memory since they will no longer be used.
	CHECK(hipFree(d_X));
	CHECK(hipFree(d_w_new));
	CHECK(hipFree(d_f_x_tilde));
	CHECK(hipFree(d_A_u));
	return;
}


/**
* The "getPredictSingleNeuronDNN_singleGPU()" global static
* function is used to apply a single GPU to calculate the prediction made by a specified single
* artificial nueron model.
*
* @param double *X - This argument will contain the pointer to a memory
*		allocated input matrix, from which the desired machine learning
*		algorithm will be calculated. THIS VARIABLE SHOULD BE ALLOCATED
* 		AND INITIALIZED BEFORE CALLING THIS FUNCTION WITH A SIZE OF "n"
* 		TIMES "m" 'DOUBLE' MEMORY SPACES.
*
* @param double *w_new - This argument will contain the pointer to a memory
*			allocated variable in which we will store the last
*			identified coefficient values for the model of a single
* 			neuron in Deep Neural Network. These coefficients will
* 			each be stored in the same row but under different
* 			columns where the first coefficient (b_0) will be stored
* 			in the column with index 0; the second coefficient (w_1)
* 			will be stored in the column index 1 and; the last
* 			coefficient (w_m) will be stored in the column index m.
* 			IT IS INDISPENSABLE THAT THIS VARIABLE IS ALLOCATED AND
* 			INITIALIZED BEFORE CALLING THIS FUNCTION WITH A VARIABLE
* 			SIZE OF "1" TIMES "m+1" 'DOUBLE' MEMORY SPACES.
* 
* @param int n - This argument will represent the total number of samples (rows)
* 		that the input matrix has, with which the output data was
*		obtained.
*
* @param int m - This argument will represent the total number of features
* 		(independent variables) that the input matrix has, with which
* 		the output data was obtained.
*
* @param int activationFunctionToBeUsed - This argument will represent the
* 					identifier of the desired activation
* 					function to be used by the neuron during
* 					its training process. Its possible valid
* 					values are the following:
*					0 = Rectified Linear Units (ReLU).
*					1 = Hyperbolic tangent (tanh).
*					2 = Logistic function.
*					3 = Raise to the 1st power.
*					4 = Raise to the 2nd power.
*					5 = Raise to the 3rd power.
*					6 = Raise to the 4th power.
*					7 = Raise to the 5th power.
*					8 = Raise to the 6th power.
*					9 = 1st order degree exponential.
*					10 = 2nd order degree exponential.
*
* @param char isClassification = This argument variable will work as a flag to
* 				indicate to the neron if it is expected from it
* 				to interpret the given data of "X" and "Y" as if
* 				their were meant for a classification problem or
* 				not. The possible valid values for this flag are
* 				the following:
*				1) "isClassification" = (int) 1 --> The neuron
* 				will interpret the data of "X" and "Y" as if they
* 				were meant for a classification problem.
*				2) "isClassification" = (int) 0 --> The neuron
* 				will interpret the data of "X" and "Y" as if they
* 				were meant for a regression problem.
*
* @param double threshold - This argument will represent desired threshold that
* 			the implementer desired the neuron to consider in
* 			classification problems. In this regard, whenever the
* 			predicted output of the neuron is higher than the
* 			defined threshold value, then that prediction should be
*			interpreted as group 1 (ussually refered to as the binary
* 			output 1). Conversely, if the predicted value is lower
* 			than the defined threshold value, then that prediction
* 			should be interpreted as group 2 (ussually refered to as
*			the binary output 0). However, have in mind that
* 			"threshold" will only be used by the neuron if the
* 			argument variable "isClassification" = 1.
*
* @param int desiredValueForGroup1 - This argument will represent the desired
*				label value to whenever an output of the neuron
* 				predicts the classification group 1. Ussually,
* 				this is label with the value of "(int) 1" but any
* 				other customized value can be assigned by the
* 				implementer. However, have in mind that this
* 				argument variable will be considered by the
* 				neuron as long as the argument variable
*				"isClassification" = 1 and only when the
*				implementer requests to the neuron a prediction
* 				through the function
* 				"predictSingleNeuronDNN_singleGPU()".
*
* @param int desiredValueForGroup2 - This argument will represent the desired
*				label value to whenever an output of the neuron
* 				predicts the classification group 2. Ussually,
* 				this is label with the value of "(int) -1" but
* 				any other customized value can be assigned by the
* 				implementer. However, have in mind that this
* 				argument variable will be considered by the
* 				neuron as long as the argument variable
*				"isClassification" = 1 and only when the
*				implementer requests to the neuron a prediction
* 				through the function
* 				"predictSingleNeuronDNN_singleGPU()".
*
* @param double *f_x_tilde - This argument will contain the pointer to a memory
* 			allocated matrix that is used to store the output of the
* 			body of the neuron in the selected GPU. IT IS
* 			INDISPENSABLE THAT THIS VARIABLE IS ALLOCATED BEFORE
* 			CALLING THIS FUNCTION WITH A VARIABLE SIZE OF "n" TIMES
* 			"1" 'DOUBLE' MEMORY SPACES.
*
* @param double *A_u - This argument will contain the pointer to a memory
* 		allocated output matrix in which the requested activation
* 		function will be applied on the argument pointer variable
* 		"f_x_tilde" and its result will be saved in "A_u". "A_u" SHOULD
* 		BE ALLOCATED BEFORE CALLING THIS FUNCTION WITH A SIZE OF "n"
* 		TIMES "p=1" 'DOUBLE' MEMORY SPACES.
* 
*
* NOTE: RESULTS ARE STORED IN "A_u".
*
* @return void
*
* @author Miranda Meza Cesar
* CREATION DATE: JANUARY 24, 2022
* LAST UPDATE: JANUARY 25, 2022
*/
__global__ static void getPredictSingleNeuronDNN_singleGPU(double *X, double *w_new, int n, int m, int activationFunctionToBeUsed, int isClassification, double threshold, int desiredValueForGroup1, int desiredValueForGroup2, double *f_x_tilde, double *A_u) {
	// We obtain the GPU thread coordinates.
	int idx = threadIdx.x + blockIdx.x * blockDim.x; // We obtain the GPU thread global coordinate.
	int tid = threadIdx.x; // We obtain the GPU thread local coordinate
	
	// If the current GPU thread is within boundary, then proceed to work with the task. Otherwise, conclude your operation.
	if (idx < n) {
		// We calculate the values of "f(x_tilde)".
		getFxTilde(X, w_new, m, f_x_tilde, tid, idx);
		
		// We calculate the currently predicted output data made by the neuron and store it in "A_u" by applying the desired activation function to "f_x_tilde".
		getActivationFunction(activationFunctionToBeUsed, f_x_tilde, A_u, idx); // We calculate A(u) and store it in the pointer variable "A_u".
		
		// Determine if the given model of a single neuron in Deep Neural Network is meant for a classification or for a regression problem to then make the predictions accordingly.
		if (isClassification == 1) {
			// We apply the threshold defined by the implementer in order to obtain a classification output and store it in "A_u".
			if (A_u[idx] > threshold) {
				A_u[idx] = desiredValueForGroup1; // Group 1 has been predicted.
			} else {
				A_u[idx] = desiredValueForGroup2; // Group 2 has been predicted.
			}
		}
	}
	
	return;
}

